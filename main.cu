#include "hip/hip_runtime.h"
// TAIGA default parameters

#define BANANA		 0		//! @param BANANA ABP: 0, banana orbits: 1
 
#define RADIONS		 1		//! @param RADIONS Real ion positions: 1, R=const 0

#define $ELM		 0		//! @param $ELM turn on <<ELM current perturbation>> mode

#define RKOLD		 0		//! @param RKOLD do not set! 0 (semi-RK: 1)

#define $FASTMODE    0//1  	//! @param $FASTMODE fastmode do not set! 0

#define $RENATE		110

#define N_BLOCKS     192		//! @param N_BLOCKS Number of blocks (max 192 on Geforce GTS450) (max 768 on Geforce GTS650Ti)
#define BLOCK_SIZE 	 1//30*4 		//! @param BLOCK_SIZE smaller is better (max 1M)

#define R_midions	 0.695		//! @param R_midions mid of ions at BANANA and no-RADIONS

#define $R_defl		2.3			//! radial position of deflection plates in meter -> TOROIDAL DEFLECTION

#define $deflH	 0				//! @param $deflH horizontal deflection in rad (up--down)  
#define $deflV	 0				//! @param $deflV vertical deflection in rad (left--right) -> TOROIDAL DEFLECTION

#define $DETPOS 0.7089 //! detector position

#if BANANA == 1
    #define $energy   0.5            // in keV
    #define $mass     2.013553212724 // in AMU (D)
    #define $diameter 50e-20         // in mm 
	#define dt		 1e-12			// timestep in seconds
	#define Nstep	 100000//00		// max step of a loop
	#define Nloop	 1000			// number of loops	
#else
	#define $energy   60				//! @param energy in keV
	#define $mass     7.016004558	//! @param mass in AMU (Li-7)
    #define $diameter 25//4/*e-20*/      //! @param diameter in mm
    #define dt       1e-9			//! @param dt timestep in seconds
    #define Nstep    2000//000			//! @param Nstep max step of a loop
    #define Nloop    1//000				//! @param Nloop number of loops

#endif



#define ERRORCHECK() cErrorCheck(__FILE__, __LINE__)
#define PI 3.141592653589792346

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#include <string.h>

#include <hip/hip_runtime_api.h>
#include "cuda/nvToolsExt.h"

#include "dataio/fieldIn.c"

#if BANANA == 1
	#include "dataio/beamInBan.c"
#elif RADIONS == 1
	#if $RENATE == 110
		#include "dataio/beamInRenate110.c"
	#else
		#include "dataio/beamIn.c"
	#endif
#else
	#include "dataio/beamInOne.c"
#endif
#include "dataio/beamOut.c"




#if RKOLD == 0
	#include "running/rk4.cu"
#else
	#include "running/rk4old.cu"
#endif

#include "running/ipol.cu"
#include "running/cyl2tor.cu"


#if RKOLD == 0
	#include "running/traj.cu"
#else
	#include "running/trajold.cu"
#endif


#include "running/ctrl.cu"

char* concat(const char *s1, const char *s2);


inline void cErrorCheck(const char *file, int line) {
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(err));
    printf(" @ %s: %d\n", file, line);
    exit(-1);
  }
}

double get_mass(char *s){
    double mass;
    
    if (strcmp(s,"D")){
        mass = 2.013553212724;
    }else if (strcmp(s,"Li")){
        mass = 7.016004558;
    }else if (strcmp(s,"Na")){
        mass = 20.0073517;
    }else if (strcmp(s,"K")){
        mass = 39.9639984821;
    }else if (strcmp(s,"H2")){
        mass = 2.013553212724;
    }else if (strcmp(s,"Li7")){
        mass = 7.016004558;
    }else if (strcmp(s,"Na20")){
        mass = 20.0073517;
    }else if (strcmp(s,"K40")){
        mass = 39.9639984821;
    }else{
        mass = (double)$mass;
    }
    
    return mass;

}

int main(int argc, char *argv[]){
    //! @param shotname name of shot folder input folder (8714,11344,11347)	
	char* shotname;
	char *beammatter;
	if (argc >= 2){
		shotname = argv[1];	
	}else{
		shotname = "11347";
	}	
	
	printf("shotname: %s\n",shotname);
	
	
	if (argc >= 3){
		beammatter = argv[2];	
	}else{
		beammatter = "Li";
	}	
	double mass = get_mass(beammatter);
	
	
	double energy=(double)$energy;
	if (argc >= 4){
		energy = atof(argv[3]);
    }
	
	double deflV=(double)$deflV;
    double deflH=(double)$deflH;
	if (argc >= 5){
		deflV = atof(argv[4]);
    }
    
    double diameter=(double)$diameter;
	if (argc >= 6){
		diameter = atof(argv[5]);
    }
		
	int NX;
	int max_blocks;
	if (argc >= 7){
		max_blocks = atoi(argv[6])/BLOCK_SIZE+1;
		//printf("max blocks: %d\n\n",max_blocks);
        //NX = atoi(argv[1]); //for the future
    }else{        
        //NX = BLOCK_SIZE*N_BLOCKS;
        max_blocks=N_BLOCKS;
	}
	
    NX = BLOCK_SIZE * max_blocks;
    

    
		
	char* folder_out=concat("results/", shotname);//! io properties folder
	// card settings
	
	
	
	int num_devices, device, max_device;
	hipGetDeviceCount(&num_devices);
	printf("Number of devices: %d\n",num_devices);
	
	if (num_devices > 1) {
        int max_multiprocessors = 0, max_device = 0;
        for (device = 0; device < num_devices; device++) {
            hipDeviceProp_t properties;
            hipGetDeviceProperties(&properties, device);
            if (max_multiprocessors < properties.multiProcessorCount) {
                max_multiprocessors = properties.multiProcessorCount;
                max_device = device;
            }
	      /*  printf("%d:%s\n",device,&properties.name);
	        printf("\tL2Cache:\t%d",	properties.l2CacheSize);
	        printf("\tNumber of cores:\t%d",	properties.warpSize);
	
	        printf("\tKernels:\t%d",	properties.concurrentKernels);
	        printf("\tThreads:\t%d",	properties.maxThreadsPerMultiProcessor);
	        printf("\tClock:\t%d",	properties.clockRate/1024);
	        printf("\n");*/
        }
        hipSetDevice(max_device);
        for (device = 0; device < num_devices; device++) {
        	if(device==max_device) printf("-->");
            hipDeviceProp_t properties;
            hipGetDeviceProperties(&properties, device);
        	printf("\t%d:\t%s\n",device,&properties.name);
        }
        
    }
	
	hipDeviceProp_t prop;
	hipGetDevice(&max_device);
	hipGetDeviceProperties( &prop, 0) ;
 /*0*/	
//	int BLOCK_SIZE = 1;//prop.maxThreadsPerBlock;
//	if(BLOCK_SIZE<1) BLOCK_SIZE=1;
//	int N_BLOCKS = 192;//0;



	// detector position
	double l_ri;

	// FOR ABP
	if (!BANANA) {
		l_ri = $DETPOS;
	// FOR BANANA ORBITS
	}else{
		l_ri = 1.0000; // just for taiga does not stop, there is no physical meaning
	}


	// phys. constants
	double eperm;

	// set timestamp

	time_t rawtime;
	struct tm *info;
	char timestamp[80];
	time( &rawtime );
  	info = localtime( &rawtime );
  	strftime(timestamp,80,"%d%b%Y_%H%M%S", info);
	
	// coords
	
	double *X_PTR[3], **x_ptr;
	double *V_PTR[3], **v_ptr;
	size_t dimXP = 3*sizeof(double*);

	double *XR,  *xr; 
	double *XZ,  *xz;
	double *XT,  *xt;

	double *VR,  *vr; 
	double *VZ,  *vz;
	double *VT,  *vt;
	
	
	
	printf("=============================\n");
	printf("Number of blocks (threads): %d\n", max_blocks);
	printf("Block size: %d\n", BLOCK_SIZE);
	printf("Number of parts: %d\n", NX);
	printf("length of a loop: %d\n", Nstep);
	printf("Number of loops: %d\n", Nloop);


	
	XR = (double*)malloc(sizeof(double)*NX);
	XZ = (double*)malloc(sizeof(double)*NX);
	XT = (double*)malloc(sizeof(double)*NX);

	VR = (double*)malloc(sizeof(double)*NX);
	VZ = (double*)malloc(sizeof(double)*NX);
	VT = (double*)malloc(sizeof(double)*NX);


	//time_t t;

	eperm = 1.60217656535e-19/1.66053892173e-27/mass;

	beamIn(XR, XZ, XT, VR, VZ, VT, energy, eperm, NX, shotname, diameter, deflH, deflV);
	/*XR[0] = 0.72;
	XZ[0] = 0.00;
	XT[0] = 0.00;*/
	

	//! position and velocity array allocation
	size_t dimX = NX * sizeof(double);

	hipMalloc((void **) &xr,  dimX); 
	hipMalloc((void **) &xz,  dimX); 
	hipMalloc((void **) &xt,  dimX); 
	hipMalloc((void **) &x_ptr,  dimXP); 

	hipMalloc((void **) &vr,  dimX); 
	hipMalloc((void **) &vz,  dimX); 
	hipMalloc((void **) &vt,  dimX); 
	hipMalloc((void **) &v_ptr,  dimXP); 



	//! coords pointers
	X_PTR[0] = xr;
	X_PTR[1] = xz;
	X_PTR[2] = xt;

	V_PTR[0] = vr;
	V_PTR[1] = vz;
	V_PTR[2] = vt;

	
	//! grid pointers
	double *G_PTR[2];
	double **g_ptr;
	size_t dimG = 2*sizeof(double*);	
	hipMalloc((void **) &g_ptr,  dimG); 
	double *RG, *rg;
	double *ZG, *zg;

	// size definitions

	//! R-grid points

	int NR = vectorReader(&RG, "input/fieldSpl", shotname, "r.spline");
	/*if ($ELM == 1){
		NR = vectorReader(&RG, "field/cuda/ipol/r.spline");
		printf("ELM mode\n");
	}else{
		NR = vectorReader(&RG, "input/fieldSpl", shotname, "r.spline");
		printf("ELM-free mode\n");
	}*/
	size_t dimR = NR * sizeof(double);
	hipMalloc((void **) &rg,  dimR); 
	
	//! Z-grid points
	int NZ = vectorReader(&ZG, "input/fieldSpl", shotname, "z.spline");
	/*if ($ELM == 1){
		NZ = vectorReader(&ZG, "field/cuda/ipol/z.spline");
	}else{
		NZ = vectorReader(&ZG, "input/fieldSpl", shotname, "z.spline");
	}*/
	//int NRZ = NR*NZ;
	size_t dimZ = NZ * sizeof(double);
	size_t dimRZ = (NR-1) * (NZ-1) * sizeof(double);
	hipMalloc((void **) &zg,  dimZ); 

   	// grid pointer
	G_PTR[0] = rg;
	G_PTR[1] = zg;


	//! MAGN. FIELD (HOST, device) ALLOCATION
	
	//!rad
	double *BR0,  *br0;  vectorReader(&BR0, "input/fieldSpl", shotname, "brad.spl11");	hipMalloc((void **) &br0,  dimRZ); 
	double *BR1,  *br1;  vectorReader(&BR1, "input/fieldSpl", shotname, "brad.spl12");	hipMalloc((void **) &br1,  dimRZ);
	double *BR2,  *br2;  vectorReader(&BR2, "input/fieldSpl", shotname, "brad.spl13");	hipMalloc((void **) &br2,  dimRZ);
	double *BR3,  *br3;  vectorReader(&BR3, "input/fieldSpl", shotname, "brad.spl14");	hipMalloc((void **) &br3,  dimRZ); 
	double *BR4,  *br4;  vectorReader(&BR4, "input/fieldSpl", shotname, "brad.spl21");	hipMalloc((void **) &br4,  dimRZ); 
	double *BR5,  *br5;  vectorReader(&BR5, "input/fieldSpl", shotname, "brad.spl22");	hipMalloc((void **) &br5,  dimRZ); 
	double *BR6,  *br6;  vectorReader(&BR6, "input/fieldSpl", shotname, "brad.spl23");	hipMalloc((void **) &br6,  dimRZ); 
	double *BR7,  *br7;  vectorReader(&BR7, "input/fieldSpl", shotname, "brad.spl24");	hipMalloc((void **) &br7,  dimRZ);
	double *BR8,  *br8;  vectorReader(&BR8, "input/fieldSpl", shotname, "brad.spl31");	hipMalloc((void **) &br8,  dimRZ); 
	double *BR9,  *br9;  vectorReader(&BR9, "input/fieldSpl", shotname, "brad.spl32");	hipMalloc((void **) &br9,  dimRZ); 
	double *BR10, *br10; vectorReader(&BR10,"input/fieldSpl", shotname, "brad.spl33");	hipMalloc((void **) &br10,  dimRZ);
	double *BR11, *br11; vectorReader(&BR11,"input/fieldSpl", shotname, "brad.spl34");	hipMalloc((void **) &br11,  dimRZ); 
	double *BR12, *br12; vectorReader(&BR12,"input/fieldSpl", shotname, "brad.spl41");	hipMalloc((void **) &br12,  dimRZ);
	double *BR13, *br13; vectorReader(&BR13,"input/fieldSpl", shotname, "brad.spl42");	hipMalloc((void **) &br13,  dimRZ); 
	double *BR14, *br14; vectorReader(&BR14,"input/fieldSpl", shotname, "brad.spl43");	hipMalloc((void **) &br14,  dimRZ); 
	double *BR15, *br15; vectorReader(&BR15,"input/fieldSpl", shotname, "brad.spl44");	hipMalloc((void **) &br15,  dimRZ);

	//!tor
	double *BT0,  *bt0;  vectorReader(&BT0, "input/fieldSpl", shotname, "btor.spl11");	hipMalloc((void **) &bt0,  dimRZ); 
	double *BT1,  *bt1;  vectorReader(&BT1, "input/fieldSpl", shotname, "btor.spl12");	hipMalloc((void **) &bt1,  dimRZ); 
	double *BT2,  *bt2;  vectorReader(&BT2, "input/fieldSpl", shotname, "btor.spl13");	hipMalloc((void **) &bt2,  dimRZ); 
	double *BT3,  *bt3;  vectorReader(&BT3, "input/fieldSpl", shotname, "btor.spl14");	hipMalloc((void **) &bt3,  dimRZ); 
	double *BT4,  *bt4;  vectorReader(&BT4, "input/fieldSpl", shotname, "btor.spl21");	hipMalloc((void **) &bt4,  dimRZ); 
	double *BT5,  *bt5;  vectorReader(&BT5, "input/fieldSpl", shotname, "btor.spl22");	hipMalloc((void **) &bt5,  dimRZ); 
	double *BT6,  *bt6;  vectorReader(&BT6, "input/fieldSpl", shotname, "btor.spl23");	hipMalloc((void **) &bt6,  dimRZ); 
	double *BT7,  *bt7;  vectorReader(&BT7, "input/fieldSpl", shotname, "btor.spl24");	hipMalloc((void **) &bt7,  dimRZ);
	double *BT8,  *bt8;  vectorReader(&BT8, "input/fieldSpl", shotname, "btor.spl31");	hipMalloc((void **) &bt8,  dimRZ); 
	double *BT9,  *bt9;  vectorReader(&BT9, "input/fieldSpl", shotname, "btor.spl32");	hipMalloc((void **) &bt9,  dimRZ); 
	double *BT10, *bt10; vectorReader(&BT10,"input/fieldSpl", shotname, "btor.spl33");	hipMalloc((void **) &bt10,  dimRZ); 
	double *BT11, *bt11; vectorReader(&BT11,"input/fieldSpl", shotname, "btor.spl34");	hipMalloc((void **) &bt11,  dimRZ); 
	double *BT12, *bt12; vectorReader(&BT12,"input/fieldSpl", shotname, "btor.spl41");	hipMalloc((void **) &bt12,  dimRZ); 
	double *BT13, *bt13; vectorReader(&BT13,"input/fieldSpl", shotname, "btor.spl42");	hipMalloc((void **) &bt13,  dimRZ);
	double *BT14, *bt14; vectorReader(&BT14,"input/fieldSpl", shotname, "btor.spl43");	hipMalloc((void **) &bt14,  dimRZ); 
	double *BT15, *bt15; vectorReader(&BT15,"input/fieldSpl", shotname, "btor.spl44");	hipMalloc((void **) &bt15,  dimRZ);
	
	//!z
	double *BZ0,  *bz0;  vectorReader(&BZ0, "input/fieldSpl", shotname, "bz.spl11");	hipMalloc((void **) &bz0,  dimRZ); 
	double *BZ1,  *bz1;  vectorReader(&BZ1, "input/fieldSpl", shotname, "bz.spl12");	hipMalloc((void **) &bz1,  dimRZ); 
	double *BZ2,  *bz2;  vectorReader(&BZ2, "input/fieldSpl", shotname, "bz.spl13");	hipMalloc((void **) &bz2,  dimRZ); 
	double *BZ3,  *bz3;  vectorReader(&BZ3, "input/fieldSpl", shotname, "bz.spl14");	hipMalloc((void **) &bz3,  dimRZ); 
	double *BZ4,  *bz4;  vectorReader(&BZ4, "input/fieldSpl", shotname, "bz.spl21");	hipMalloc((void **) &bz4,  dimRZ);
	double *BZ5,  *bz5;  vectorReader(&BZ5, "input/fieldSpl", shotname, "bz.spl22");	hipMalloc((void **) &bz5,  dimRZ); 
	double *BZ6,  *bz6;  vectorReader(&BZ6, "input/fieldSpl", shotname, "bz.spl23");	hipMalloc((void **) &bz6,  dimRZ);
	double *BZ7,  *bz7;  vectorReader(&BZ7, "input/fieldSpl", shotname, "bz.spl24");	hipMalloc((void **) &bz7,  dimRZ);
	double *BZ8,  *bz8;  vectorReader(&BZ8, "input/fieldSpl", shotname, "bz.spl31");	hipMalloc((void **) &bz8,  dimRZ); 
	double *BZ9,  *bz9;  vectorReader(&BZ9, "input/fieldSpl", shotname, "bz.spl32");	hipMalloc((void **) &bz9,  dimRZ); 
	double *BZ10, *bz10; vectorReader(&BZ10,"input/fieldSpl", shotname, "bz.spl33");	hipMalloc((void **) &bz10,  dimRZ); 
	double *BZ11, *bz11; vectorReader(&BZ11,"input/fieldSpl", shotname, "bz.spl34");	hipMalloc((void **) &bz11,  dimRZ); 
	double *BZ12, *bz12; vectorReader(&BZ12,"input/fieldSpl", shotname, "bz.spl41");	hipMalloc((void **) &bz12,  dimRZ);
	double *BZ13, *bz13; vectorReader(&BZ13,"input/fieldSpl", shotname, "bz.spl42");	hipMalloc((void **) &bz13,  dimRZ); 
	double *BZ14, *bz14; vectorReader(&BZ14,"input/fieldSpl", shotname, "bz.spl43");	hipMalloc((void **) &bz14,  dimRZ);
	double *BZ15, *bz15; vectorReader(&BZ15,"input/fieldSpl", shotname, "bz.spl44");	hipMalloc((void **) &bz15,  dimRZ);
	

	// magnetic field pointer array
	// magnetic field (HOST, device)
	size_t dimB = 16*sizeof(double*);
		
	double *BR_PTR[16];	double **br_ptr;	
	hipMalloc((void **) &br_ptr,  dimB); 
	double *BT_PTR[16];	double **bt_ptr;
	hipMalloc((void **) &bt_ptr,  dimB); 
	double *BZ_PTR[16];	double **bz_ptr;
	hipMalloc((void **) &bz_ptr,  dimB); 	
		
	
	
	//! MAGN. FIELD POINTERS
	
	//!rad	
	BR_PTR[0] = br0;	BR_PTR[1] = br1;	BR_PTR[2] = br2;	BR_PTR[3] = br3;
	BR_PTR[4] = br4;	BR_PTR[5] = br5;	BR_PTR[6] = br6;	BR_PTR[7] = br7;
	BR_PTR[8] = br8;	BR_PTR[9] = br9;	BR_PTR[10] = br10;	BR_PTR[11] = br11;
	BR_PTR[12] = br12;	BR_PTR[13] = br13;	BR_PTR[14] = br14;	BR_PTR[15] = br15;
	
	//!tor
	BT_PTR[0] = bt0;	BT_PTR[1] = bt1;	BT_PTR[2] = bt2;	BT_PTR[3] = bt3;
	BT_PTR[4] = bt4;	BT_PTR[5] = bt5;	BT_PTR[6] = bt6;	BT_PTR[7] = bt7;
	BT_PTR[8] = bt8;	BT_PTR[9] = bt9;	BT_PTR[10] = bt10;	BT_PTR[11] = bt11;
	BT_PTR[12] = bt12;	BT_PTR[13] = bt13;	BT_PTR[14] = bt14;	BT_PTR[15] = bt15;
	
	//!z
	BZ_PTR[0] = bz0;	BZ_PTR[1] = bz1;	BZ_PTR[2] = bz2;	BZ_PTR[3] = bz3;
	BZ_PTR[4] = bz4;	BZ_PTR[5] = bz5;	BZ_PTR[6] = bz6;	BZ_PTR[7] = bz7;
	BZ_PTR[8] = bz8;	BZ_PTR[9] = bz9;	BZ_PTR[10] = bz10;	BZ_PTR[11] = bz11;
	BZ_PTR[12] = bz12;	BZ_PTR[13] = bz13;	BZ_PTR[14] = bz14;	BZ_PTR[15] = bz15;
	
	
	
		
	// temporary test data
	double *TMP, *tmp;
	TMP = (double *)malloc(dimR);
	hipMalloc((void **) &tmp,  dimR); 
	
	
		
	// field direction on first
	double *BD1, *bd1;
	double *BD2, *bd2;
	BD1 = (double *)malloc(dimX);
	BD2 = (double *)malloc(dimX);
	hipMalloc((void **) &bd1,  dimX); 
	hipMalloc((void **) &bd2,  dimX); 
	
	

	//! CUDA profiler START
	hipProfilerStart();
	
	//! MEMCOPY (HOST2device)
	
	


	//! GRID COORDS	
	hipMemcpy(rg, RG, dimR, hipMemcpyHostToDevice);
	hipMemcpy(zg, ZG, dimZ, hipMemcpyHostToDevice);
	hipMemcpy(g_ptr, G_PTR, dimG, hipMemcpyHostToDevice);
	
	//! MAGNETIC FIELD
	
	//!rad	
	hipMemcpy(br0, BR0, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br1, BR1, dimRZ, hipMemcpyHostToDevice);	
	hipMemcpy(br2, BR2, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br3, BR3, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br4, BR4, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br5, BR5, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br6, BR6, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br7, BR7, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br8, BR8, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br9, BR9, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br10, BR10, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br11, BR11, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br12, BR12, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br13, BR13, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br14, BR14, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(br15, BR15, dimRZ, hipMemcpyHostToDevice);			
	hipMemcpy(br_ptr, BR_PTR, dimB, hipMemcpyHostToDevice);
	
	//!tor
	hipMemcpy(bt0, BT0, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt1, BT1, dimRZ, hipMemcpyHostToDevice);	
	hipMemcpy(bt2, BT2, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt3, BT3, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt4, BT4, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt5, BT5, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt6, BT6, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt7, BT7, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt8, BT8, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt9, BT9, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt10, BT10, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt11, BT11, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt12, BT12, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt13, BT13, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt14, BT14, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bt15, BT15, dimRZ, hipMemcpyHostToDevice);			
	hipMemcpy(bt_ptr, BT_PTR, dimB, hipMemcpyHostToDevice);
	
	//!z
	hipMemcpy(bz0, BZ0, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz1, BZ1, dimRZ, hipMemcpyHostToDevice);	
	hipMemcpy(bz2, BZ2, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz3, BZ3, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz4, BZ4, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz5, BZ5, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz6, BZ6, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz7, BZ7, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz8, BZ8, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz9, BZ9, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz10, BZ10, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz11, BZ11, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz12, BZ12, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz13, BZ13, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz14, BZ14, dimRZ, hipMemcpyHostToDevice);
	hipMemcpy(bz15, BZ15, dimRZ, hipMemcpyHostToDevice);			
	hipMemcpy(bz_ptr, BZ_PTR, dimB, hipMemcpyHostToDevice);



	//! ION COORDS (HOST2device)
	/*hipMemcpy(xr, XR, dimX, hipMemcpyHostToDevice);
	hipMemcpy(xz, XZ, dimX, hipMemcpyHostToDevice);
	hipMemcpy(xt, XT, dimX, hipMemcpyHostToDevice);*/
	hipMemcpy(x_ptr, X_PTR, dimXP, hipMemcpyHostToDevice);	

	//! ION SPEEDS (HOST2device)
	/*hipMemcpy(vr, VR, dimX, hipMemcpyHostToDevice);
	hipMemcpy(vz, VZ, dimX, hipMemcpyHostToDevice);
	hipMemcpy(vt, VT, dimX, hipMemcpyHostToDevice);*/
	hipMemcpy(v_ptr, V_PTR, dimXP, hipMemcpyHostToDevice);

	
	// EXECUTION	




	addData1(XR,NX,folder_out,timestamp,"t_rad.dat");
	addData1(XZ,NX,folder_out,timestamp,"t_z.dat");
	addData1(XZ,NX,folder_out,timestamp,"t_tor.dat");

	//! Set CUDA timer 
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	printf("ionV:  0.\t %lf\t %lf\t %lf\n",VR[0],VZ[0],VT[0]);
	printf("ionX:  0.\t %lf\t %lf\t %lf\n",XR[0],XZ[0],XT[0]);
	printf("ionX:  1.\t %lf\t %lf\t %lf\n",XR[1],XZ[1],XT[1]);
	
	
	// BANANA
	if (BANANA==1){
		printf("BANANA CTRL\n");
		// ION COORDS (HOST2device)
		hipMemcpy(xr, XR, dimX, hipMemcpyHostToDevice);
		hipMemcpy(xz, XZ, dimX, hipMemcpyHostToDevice);
		hipMemcpy(xt, XT, dimX, hipMemcpyHostToDevice);
		//hipMemcpy(x_ptr, X_PTR, dimXP, hipMemcpyHostToDevice);	

		// ION SPEEDS (HOST2device)
		hipMemcpy(vr, VR, dimX, hipMemcpyHostToDevice);
		hipMemcpy(vz, VZ, dimX, hipMemcpyHostToDevice);
		hipMemcpy(vt, VT, dimX, hipMemcpyHostToDevice);
				
		banCtrl <<< max_blocks, BLOCK_SIZE >>> (NR,NZ,br_ptr,bz_ptr,bt_ptr,g_ptr,x_ptr,bd1,bd2);
		hipMemcpy(BD1, bd1, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(BD2, bd2, dimX, hipMemcpyDeviceToHost);
		addData1(BD1,NX,folder_out,timestamp,"d_b1.dat");
		addData1(BD2,NX,folder_out,timestamp,"d_b2.dat");
		addData1(VR,NX,folder_out,timestamp,"d_vr.dat");
		addData1(VZ,NX,folder_out,timestamp,"d_vz.dat");
		addData1(VT,NX,folder_out,timestamp,"d_vt.dat");
	}	
	
	for (int step_i=0;step_i<Nloop;step_i++){

		
		if ($FASTMODE==0){
			// ION COORDS (HOST2device)
			hipMemcpy(xr, XR, dimX, hipMemcpyHostToDevice);
			hipMemcpy(xz, XZ, dimX, hipMemcpyHostToDevice);
			hipMemcpy(xt, XT, dimX, hipMemcpyHostToDevice);
			//hipMemcpy(x_ptr, X_PTR, dimXP, hipMemcpyHostToDevice);	

			// ION SPEEDS (HOST2device)
			hipMemcpy(vr, VR, dimX, hipMemcpyHostToDevice);
			hipMemcpy(vz, VZ, dimX, hipMemcpyHostToDevice);
			hipMemcpy(vt, VT, dimX, hipMemcpyHostToDevice);
			//hipMemcpy(v_ptr, V_PTR, dimXP, hipMemcpyHostToDevice);
		}else{
			double *PROFR, *PROFD;
			int N_beamdens = vectorReader0(&PROFR,"dataio/data/prof_r.dat");
			vectorReader0(&PROFD,"dataio/data/prof_d.dat");
		}
		
		// CUDA CODE, timer and Error catch	
		//ERRORCHECK();
		hipEventRecord(start, 0);
		ctrl <<< max_blocks, BLOCK_SIZE >>> (NR,NZ,br_ptr,bz_ptr,bt_ptr,g_ptr,x_ptr,v_ptr,tmp,eperm,l_ri);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		ERRORCHECK();

		// ION COORDS (device2HOST)
		hipMemcpy(XR, xr, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(XZ, xz, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(XT, xt, dimX, hipMemcpyDeviceToHost);
		//ERRORCHECK();
		// ION SPEEDS (device2HOST)
		hipMemcpy(VR, vr, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(VZ, vz, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(VT, vt, dimX, hipMemcpyDeviceToHost);
		//ERRORCHECK();
		// Save data to files
		printf("Step\t%d/%d\n",step_i,Nloop);
		addData1(XR,NX,folder_out,timestamp,"t_rad.dat");
		addData1(XZ,NX,folder_out,timestamp,"t_z.dat");
		addData1(XZ,NX,folder_out,timestamp,"t_tor.dat");
		

		

	//printf("ion:  0.\t %lf\t %lf\t %lf\n",XR[0],XZ[0],XT[0]);
	//printf("ion:  1.\t %lf\t %lf\t %lf\n",XR[1],XZ[1],XT[1]);
	//printf("ion:  0.\t %lf\t %lf\t %lf\n",VR[0],VZ[0],VT[0]);
	}	
	
	// Get CUDA timer 
	hipEventElapsedTime(&time, start, stop);
	printf ("Time for the kernel: %f s\n", time/1000.0);

/*	// ION COORDS (device2HOST)
	hipMemcpy(XR, xr, dimX, hipMemcpyDeviceToHost);
	hipMemcpy(XZ, xz, dimX, hipMemcpyDeviceToHost);
	hipMemcpy(XT, xt, dimX, hipMemcpyDeviceToHost);

	// ION SPEEDS (device2HOST)
	hipMemcpy(VR, vr, dimX, hipMemcpyDeviceToHost);
	hipMemcpy(VZ, vz, dimX, hipMemcpyDeviceToHost);
	hipMemcpy(VT, vt, dimX, hipMemcpyDeviceToHost);*/

	//! MEMCOPY (device2HOST)
	hipMemcpy(TMP, tmp, dimR, hipMemcpyDeviceToHost);
	if(TMP[0]!=42.24){
		printf("\n+---	-------------------+\n | Fatal error in running. | \n | The CUDA did not run well. |\n+-----------------------+\n");
	}else{
		printf("\n	Memcopy OK.\n");
	}
	




	//! CUDA profiler STOP
	hipProfilerStop();
	/*
	printf("ion:  0.\t %18.18le\t %18.18le\t %18.18le\n",VR[0],VZ[0],VT[0]);
	printf("ion:  0.\t %18.18le\t %18.18le\t %18.18le\n",XR[0],XZ[0],XT[0]);
	printf("----------------------------------------------------------\n");
*/

	printf("----------------------------------------------------------\n");
	printf("ion:  0.\t %lf\t %lf\t %lf\n",XR[0],XZ[0],XT[0]);
	printf("----------------------------------------------------------\n");
	for(int i=1; i<20; i++){
		printf("ion: %2d.\t %le\t %le\t %le\n",i,XR[i],XZ[i],XT[i]);
	}
	printf("----------------------------------------------------------\n");
/*
	//printf("ion:  0.\t %18.18le\t %18.18le\t %18.18le\n",XR[0],XZ[0],XT[0]);
	//printf("ion:  0.\t %18.18le\t %18.18le\t %18.18le\n",VR[0],VZ[0],VT[0]);
	
	*/
	//! Save data to files
	saveData1(XR,NX,folder_out,timestamp,"rad.dat");
	saveData1(XZ,NX,folder_out,timestamp,"z.dat");
	saveData1(XT,NX,folder_out,timestamp,"tor.dat");
	
	
	
	saveDataHT(concat("Shot ID: ",shotname),folder_out,timestamp);
	saveDataHT(concat("Run ID:  ",timestamp),folder_out,timestamp);
	saveDataHT("-----------------------------------",folder_out,timestamp);
	if(BANANA){
		saveDataHT("BANANA ORBITS",folder_out,timestamp);
	}else{		
		saveDataHT("ABP ION TRAJECTORIES",folder_out,timestamp);
		if(RADIONS){
			saveDataHT("(Real ionization position)",folder_out,timestamp);
			if($RENATE==110){
				saveDataHT("(TS + Renate 1.1.0)",folder_out,timestamp);
			}
		}else{
			saveDataHT("(R=const ionization)",folder_out,timestamp);
		}
	}
	saveDataHT("-----------------------------------",folder_out,timestamp);
	saveDataH("Beam energy","keV",energy,folder_out,timestamp);
	saveDataH("Atomic mass","AMU",mass,folder_out,timestamp);
	saveDataH("Beam diameter","mm",diameter,folder_out,timestamp);
	saveDataH2("Deflation (H/V)","°",$deflH,$deflV,folder_out,timestamp);
	if(!RADIONS&&!BANANA){	
		saveDataH("Ion. position (R)","m",R_midions,folder_out,timestamp);
	}
	
	saveDataH("Number of ions","",NX,folder_out,timestamp);
	saveDataHT("-----------------------------------",folder_out,timestamp);
	
	saveDataH("Detector position (R)","m",l_ri,folder_out,timestamp);
	
	saveDataHT("-----------------------------------",folder_out,timestamp);
	
	saveDataH("Timestep","s",dt,folder_out,timestamp);
	
	
	saveDataHT("-----------------------------------",folder_out,timestamp);
	
	saveDataH("Kernel runtime", "s", time/1000.0,folder_out,timestamp);
	saveDataHT("-----------------------------------",folder_out,timestamp);
	saveDataH("Number of blocks (threads)", "", max_blocks,folder_out,timestamp);
	saveDataH("Block size", "", BLOCK_SIZE,folder_out,timestamp);
	saveDataH("Length of a loop", "", Nstep,folder_out,timestamp);
	saveDataH("Number of loops", "", Nloop,folder_out,timestamp);
	
	

	printf("\nData folder: %s/%s\n\n",folder_out,timestamp);



	//! Free CUDA

	hipFree(x_ptr);	hipFree(xr);	hipFree(xz);	hipFree(xt);

	hipFree(g_ptr);	hipFree(rg);	hipFree(zg);	
	
	hipFree(br_ptr);	hipFree(bz_ptr);	hipFree(bt_ptr);	
	
	hipFree(br0);	hipFree(br1);	hipFree(br2);	hipFree(br3);	
	hipFree(br4);	hipFree(br5);	hipFree(br6);	hipFree(br7);	
	hipFree(br8);	hipFree(br9);	hipFree(br10);	hipFree(br11);	
	hipFree(br12);	hipFree(br13);	hipFree(br14);	hipFree(br15);
		
	hipFree(bz0);	hipFree(bz1);	hipFree(bz2);	hipFree(bz3);
	hipFree(bz4);	hipFree(bz5);	hipFree(bz6);	hipFree(bz7);
	hipFree(bz8);	hipFree(bz9);	hipFree(bz10);	hipFree(bz11);
	hipFree(bz12);	hipFree(bz13);	hipFree(bz14);	hipFree(bz15);
	
	hipFree(bt0);	hipFree(bt1);	hipFree(bt2);	hipFree(bt3);	
	hipFree(bt4);	hipFree(bt5);	hipFree(bt6);	hipFree(bt7);	
	hipFree(bt8);	hipFree(bt9);	hipFree(bt10);	hipFree(bt11);	
	hipFree(bt12);	hipFree(bt13);	hipFree(bt14);	hipFree(bt15);	


	//! Free RAM
	free(RG);	free(ZG);	


	free(XR);	free(XZ);	free(XT);
	//	free(G_PTR);
	//	free(BR_PTR);	free(BZ_PTR);	free(BT_PTR);	
	
	free(BR0);	free(BR1);	free(BR2);	free(BR3);
	free(BR4);	free(BR5);	free(BR6);	free(BR7);	
	free(BR8);	free(BR9);	free(BR10);	free(BR11);	
	free(BR12);	free(BR13);	free(BR14);	free(BR15);
		
	free(BZ0);	free(BZ1);	free(BZ2);	free(BZ3);	
	free(BZ4);	free(BZ5);	free(BZ6);	free(BZ7);	
	free(BZ8);	free(BZ9);	free(BZ10);	free(BZ11);	
	free(BZ12);	free(BZ13);	free(BZ14);	free(BZ15);
	
	free(BT0);	free(BT1);	free(BT2);	free(BT3);
	free(BT4);	free(BT5);	free(BT6);	free(BT7);
	free(BT8);	free(BT9);	free(BT10);	free(BT11);
	free(BT12);	free(BT13);	free(BT14);	free(BT15);	
	
	
	//! FREE TMP variables (RAM, cuda)
	free(TMP);	hipFree(tmp);

	printf("Ready.\n\n");
}

char* concat(const char *s1, const char *s2){
    char *result = (char*)malloc(strlen(s1)+strlen(s2)+1);//+1 for the zero-terminator
    //in real code you would check for errors in malloc here
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}
