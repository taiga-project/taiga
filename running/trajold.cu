#include "hip/hip_runtime.h"

//double gyok2;

__device__ /*double int*/ void copyLocal(double *rg, int NR, double *zg, int NZ, double l_r, double l_z, int *rzci, double *lp_br, double *lp_bz, double *lp_bt,  double **br_ptr, double **bz_ptr, double **bt_ptr){
	int rci, zci;
	int i, i2;
	
	for(rci=0;(rg[rci+1]<l_r)&&(rci<NR-1);rci++){;}
	
	
	for(zci=0;(zg[zci+1]<l_z)&&(zci<NR-1);zci++){;}
	
	
	// Particle leave out the cell
	if ((rzci[0]!=rci)||(rzci[1]!=zci)){
		rzci[0]=rci; 	
		rzci[1]=zci; 	
	
		for(i=0;i<16;i++){
	
			i2 = (rzci[0])*(NZ-1)+rzci[1];
		
			lp_br[i]=br_ptr[i][i2];			
			lp_bz[i]=bz_ptr[i][i2];			
			lp_bt[i]=bt_ptr[i][i2];
		
		}
	}
	
	//return i2;	
}


__device__ double localField(double *lp_b, double dr, double dz){

/* MATLAB CODE:
    sample2(3) =c11(bs1,bs2)*dsx^3*dsy^3 + c12(bs1,bs2)*dsx^3*dsy^2 + c13(bs1,bs2)*dsx^3*dsy + c14(bs1,bs2)*dsx^3 + ...
                c21(bs1,bs2)*dsx^2*dsy^3 + c22(bs1,bs2)*dsx^3*dsy^2 + c23(bs1,bs2)*dsx^2*dsy + c24(bs1,bs2)*dsx^2 + ...
                c31(bs1,bs2)*dsx  *dsy^3 + c32(bs1,bs2)*dsx  *dsy^2 + c33(bs1,bs2)*dsx  *dsy + c34(bs1,bs2)*dsx    + ...
                c41(bs1,bs2)      *dsy^3 + c42(bs1,bs2)      *dsy^2 + c43(bs1,bs2)      *dsy + c44(bs1,bs2);*/
                
                
    double blocal = 0.0, tmp[16] ;
    for(int i=0;i<4;i++){
 	    for(int j=0;j<4;j++){
		    tmp[i*4+j] = lp_b[i*4+j]*pow(dr,3-i)*pow(dz,3-j);
			
        }
    }   
    
   	for(int i=0;i<4;i++){
    	for(int j=0;j<4;j++){
		    blocal+=tmp[i*4+j];
        }
    }   
    
	return blocal;
}

/*

l_x local coordinates

*/
__device__ void traj(double *rg, int NR, double *zg, int NZ, double *l_x, double *l_v, double **br_ptr, double **bz_ptr, double **bt_ptr, double eperm, double l_ri){

	// next grid
	int rzci[2];
	rzci[0]=-1;
	rzci[1]=-1;
		
	double lp_br[16];
	double lp_bz[16];
	double lp_bt[16];	
	
	double l_br=0,l_bz,l_bt;
	double dr,dz;
	double drT, l_rT;
	
	double l_vr, l_vz, l_vt, l_vor, l_voz, l_vot;
	double l_r=l_x[0], l_z=l_x[1], l_t=l_x[2];
	double l_or, l_oz, l_ot;
	
	int finished = 0;	
	
	// Temporary data -- from here
	
	l_vr =  l_v[0];
	l_vz =  l_v[1];
	l_vt =  l_v[2];

	// Temp. data -- until here	

	//double l_ri = /*0.725*/ 0.7089;
	
	int loopi;
	for (loopi=0;(loopi<Nstep && !finished);loopi++){
		// Get local magnetic field	

		l_rT = cyl2tor_coord(l_r, l_t);
		copyLocal(rg,NR,zg,NZ,l_rT,l_z,rzci,lp_br,lp_bz,lp_bt,br_ptr,bz_ptr,bt_ptr);	
		
		//dr = l_r-rg[rzci[0]];
		dr = l_rT-rg[rzci[0]];
		dz = l_z-zg[rzci[1]];
	
		l_br =  localField(lp_br,dr,dz);
		l_bz =  localField(lp_bz,dr,dz);
		l_bt =  localField(lp_bt,dr,dz);

		l_br = cyl2tor_rad(l_br, l_bt, l_r, l_t);
		l_bt = cyl2tor_tor(l_br, l_bt, l_r, l_t);
	

		// archivate coordinates
		l_or  = l_r;	l_oz  = l_z;	l_ot  = l_t;
		l_vor = l_vr;	l_voz = l_vz;	l_vot = l_vt;
	
		// RK4 for speed
		l_vr = rk4r(l_vor, l_voz, l_vot, l_br, l_bz, l_bt, eperm);
		l_vz = rk4z(l_vor, l_voz, l_vot, l_br, l_bz, l_bt, eperm);
		l_vt = rk4t(l_vor, l_voz, l_vot, l_br, l_bz, l_bt, eperm);
	
		// new coordinates
		l_r = l_or + dt*(l_vr+l_vor)/2;
		l_z = l_oz + dt*(l_vz+l_voz)/2;
		l_t = l_ot + dt*(l_vt+l_vot)/2;
	
		// finished? (interpolation)
		finished = ipol(l_r, l_z, l_t, l_or, l_oz, l_ot, l_ri, l_x, l_vr);

	}

	
	l_v[0] = l_vr;
	l_v[1] = l_vz;
	l_v[2] = l_vt;
	

	if (!finished){
		l_x[0] = l_r;
		l_x[1] = l_z;
		l_x[2] = l_t;
	}
	
	

	//return /*sqrt(1000*energy*eperm*2);//*/(double)loopi;
}
