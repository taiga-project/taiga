#include <hip/hip_runtime.h>

void sync_device_structs(TaigaGlobals *g_device, TaigaGlobals *g_shared, TaigaCommons *c_device, TaigaCommons *c_shared){
    hipMemcpy(c_device, c_shared, sizeof(TaigaCommons), hipMemcpyHostToDevice);
    if (!FASTMODE){
        hipMemcpy(g_device, g_shared, sizeof(TaigaGlobals), hipMemcpyHostToDevice);
    }
}

void coord_memcopy_back(BeamProp beam, ShotProp shot, RunProp run, TaigaGlobals *g_host, TaigaGlobals *g_shared){
    size_t size_coord = run.block_size * run.block_number * sizeof(double);
    size_t size_detcellid = run.block_size * run.block_number * sizeof(int);

    double* host_rad =(double*)malloc(size_coord);
    double* host_z =(double*)malloc(size_coord);
    double* host_tor =(double*)malloc(size_coord);
    double* host_vrad =(double*)malloc(size_coord);
    double* host_vz =(double*)malloc(size_coord);
    double* host_vtor =(double*)malloc(size_coord);
    int* host_detcellid =(int*)malloc(size_detcellid);

    hipMemcpy(host_rad,  g_shared->rad,  size_coord, hipMemcpyDeviceToHost); g_host->rad = host_rad;
    hipMemcpy(host_z,    g_shared->z,    size_coord, hipMemcpyDeviceToHost); g_host->z = host_z;
    hipMemcpy(host_tor,  g_shared->tor,  size_coord, hipMemcpyDeviceToHost); g_host->tor = host_tor;
    hipMemcpy(host_vrad, g_shared->vrad, size_coord, hipMemcpyDeviceToHost); g_host->vrad = host_vrad;
    hipMemcpy(host_vz,   g_shared->vz,   size_coord, hipMemcpyDeviceToHost); g_host->vz = host_vz;
    hipMemcpy(host_vtor, g_shared->vtor, size_coord, hipMemcpyDeviceToHost); g_host->vtor = host_vtor;
    hipMemcpy(host_detcellid, g_shared->detcellid, size_detcellid, hipMemcpyDeviceToHost); g_host->detcellid = host_detcellid;
}