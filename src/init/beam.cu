#include <hip/hip_runtime.h>
#include "dataio/beam.h"

void init_coords(BeamProp *beam, ShotProp *shot, RunProp *run, TaigaGlobals *g_host, TaigaGlobals *g_shared) {
    size_t size_coord = run->block_size * run->block_number * sizeof(double);
    size_t size_detcellid = run->block_size * run->block_number * sizeof(int);
    size_t size_globals = sizeof(TaigaGlobals);

    double* shared_rad;
    double* shared_z;
    double* shared_tor;
    double* shared_vrad;
    double* shared_vz;
    double* shared_vtor;
    int* shared_detcellid;

    if (!FASTMODE){
        g_host->rad = (double*)malloc(size_coord);
        g_host->z   = (double*)malloc(size_coord);
        g_host->tor = (double*)malloc(size_coord);
        g_host->vrad = (double*)malloc(size_coord);
        g_host->vz   = (double*)malloc(size_coord);
        g_host->vtor = (double*)malloc(size_coord);
        g_host->detcellid = (int*)malloc(size_detcellid);
        load_beam(g_host, beam, shot, run);

        for (int i=0; i<run->block_size * run->block_number; ++i){
            g_host->detcellid[i] = CALCULATION_NOT_FINISHED;
        }

        memcpy(g_shared, g_host, size_globals);
    }

    hipMalloc((void **) &shared_rad, size_coord);
    hipMalloc((void **) &shared_z,   size_coord);
    hipMalloc((void **) &shared_tor, size_coord);
    hipMalloc((void **) &shared_vrad, size_coord);
    hipMalloc((void **) &shared_vz,   size_coord);
    hipMalloc((void **) &shared_vtor, size_coord);
    hipMalloc((void **) &shared_detcellid, size_detcellid);

    if (!FASTMODE){
        hipMemcpy(shared_rad,       g_host->rad,       size_coord,  hipMemcpyHostToDevice);
        hipMemcpy(shared_z,         g_host->z,         size_coord,  hipMemcpyHostToDevice);
        hipMemcpy(shared_tor,       g_host->tor,       size_coord,  hipMemcpyHostToDevice);
        hipMemcpy(shared_vrad,      g_host->vrad,      size_coord,  hipMemcpyHostToDevice);
        hipMemcpy(shared_vz,        g_host->vz,        size_coord,  hipMemcpyHostToDevice);
        hipMemcpy(shared_vtor,      g_host->vtor,      size_coord,  hipMemcpyHostToDevice);
        hipMemcpy(shared_detcellid, g_host->detcellid, size_detcellid, hipMemcpyHostToDevice);
    }

    g_shared->rad  = shared_rad;
    g_shared->z    = shared_z;
    g_shared->tor  = shared_tor;
    g_shared->vrad = shared_vrad;
    g_shared->vz   = shared_vz;
    g_shared->vtor = shared_vtor;
    g_shared->detcellid = shared_detcellid;
}

void init_beam_profile(BeamProfile *device_prof, ShotProp shot){
    BeamProfile *host_prof, *shared_prof;
    size_t size_prof = sizeof(BeamProfile);
    host_prof = (BeamProfile*)malloc(size_prof);
    shared_prof = (BeamProfile*)malloc(size_prof);
    init_ion_profile(shot.name, host_prof);
    size_t size_rad_prof = sizeof(double)*host_prof->radial_length;
    size_t size_cross_prof = sizeof(double)*host_prof->cross_length;
    
    double *shared_radial_grid;
    double *shared_radial_profile;
    double *shared_cross_grid;
    double *shared_cross_profile;
    
    hipMalloc((void **) &shared_radial_grid,    size_rad_prof);
    hipMalloc((void **) &shared_radial_profile, size_rad_prof);
    hipMalloc((void **) &shared_cross_grid,     size_cross_prof);
    hipMalloc((void **) &shared_cross_profile,  size_cross_prof);
    
    hipMemcpy(shared_radial_grid,    host_prof->radial_grid,    size_rad_prof, hipMemcpyHostToDevice);
    hipMemcpy(shared_radial_profile, host_prof->radial_profile, size_rad_prof, hipMemcpyHostToDevice);
    hipMemcpy(shared_cross_grid,     host_prof->cross_grid,     size_cross_prof, hipMemcpyHostToDevice);
    hipMemcpy(shared_cross_profile,  host_prof->cross_profile,  size_cross_prof, hipMemcpyHostToDevice);
    
    shared_prof->radial_grid    = shared_radial_grid;
    shared_prof->radial_profile = shared_radial_profile;
    shared_prof->cross_grid     = shared_cross_grid;
    shared_prof->cross_profile  = shared_cross_profile;
    
    hipMemcpy(device_prof,  shared_prof,  size_prof, hipMemcpyHostToDevice);
}
