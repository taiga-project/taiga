#include "hip/hip_runtime.h"
#include "utils/cuda.cuh"

void sync_device_structs(TaigaGlobals *g_device, TaigaGlobals *g_shared, TaigaCommons *c_device, TaigaCommons *c_shared,
                         bool is_all_io){
    CHECK_ERROR(hipMemcpy(c_device, c_shared, sizeof(TaigaCommons), hipMemcpyHostToDevice));
    if (is_all_io){
        CHECK_ERROR(hipMemcpy(g_device, g_shared, sizeof(TaigaGlobals), hipMemcpyHostToDevice));
    }
}

void coord_memcopy_back(BeamProp beam, ShotProp shot, RunProp run, TaigaGlobals *g_host, TaigaGlobals *g_shared){
    size_t size_coord = run.block_size * run.block_number * sizeof(double);
    size_t size_detcellid = run.block_size * run.block_number * sizeof(int);

    double* host_rad =(double*)malloc(size_coord);
    double* host_z =(double*)malloc(size_coord);
    double* host_tor =(double*)malloc(size_coord);
    double* host_vrad =(double*)malloc(size_coord);
    double* host_vz =(double*)malloc(size_coord);
    double* host_vtor =(double*)malloc(size_coord);
    double* host_intensity =(double*)malloc(size_coord);
    double* host_time_of_flight =(double*)malloc(size_coord);
    int* host_detcellid =(int*)malloc(size_detcellid);

    CHECK_ERROR(hipMemcpy(host_rad,  g_shared->rad,  size_coord, hipMemcpyDeviceToHost));
    g_host->rad = host_rad;
    CHECK_ERROR(hipMemcpy(host_z,    g_shared->z,    size_coord, hipMemcpyDeviceToHost));
    g_host->z = host_z;
    CHECK_ERROR(hipMemcpy(host_tor,  g_shared->tor,  size_coord, hipMemcpyDeviceToHost));
    g_host->tor = host_tor;
    CHECK_ERROR(hipMemcpy(host_vrad, g_shared->vrad, size_coord, hipMemcpyDeviceToHost));
    g_host->vrad = host_vrad;
    CHECK_ERROR(hipMemcpy(host_vz,   g_shared->vz,   size_coord, hipMemcpyDeviceToHost));
    g_host->vz = host_vz;
    CHECK_ERROR(hipMemcpy(host_vtor, g_shared->vtor, size_coord, hipMemcpyDeviceToHost));
    g_host->vtor = host_vtor;
    CHECK_ERROR(hipMemcpy(host_intensity, g_shared->intensity, size_coord, hipMemcpyDeviceToHost));
    g_host->intensity = host_intensity;
    CHECK_ERROR(hipMemcpy(host_time_of_flight, g_shared->time_of_flight, size_coord, hipMemcpyDeviceToHost));
    g_host->time_of_flight = host_time_of_flight;
    CHECK_ERROR(hipMemcpy(host_detcellid, g_shared->detcellid, size_detcellid, hipMemcpyDeviceToHost));
    g_host->detcellid = host_detcellid;
}