#include <hip/hip_runtime.h>
#include "core/generate_coords.cuh"
#include "utils/physics.h"

void init_fastmode(BeamProp beam, ShotProp shot, RunProp run, TaigaGlobals *device_global){
    BeamProfile *device_prof;
    size_t size_prof = sizeof(BeamProfile);
    hipMalloc((void **) &device_prof, size_prof);
    init_beam_profile(device_prof, shot);
    generate_coords <<< run.block_number, run.block_size >>> (device_global, beam, device_prof, get_mass(beam.species, beam.charge));
}