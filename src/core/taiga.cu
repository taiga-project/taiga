#include "hip/hip_runtime.h"
#include "init_beamlet.cuh"

__global__ void taiga(TaigaGlobals *g, TaigaCommons *c, double *service_var){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;    // thread index
    
    if (g->detcellid[idx] == -1){
        double generalised_coordinates[X_SIZE];
        int detcellid = g->detcellid[idx];
        
        generalised_coordinates[0] = g->rad[idx];
        generalised_coordinates[1] = g->z[idx];
        generalised_coordinates[2] = g->tor[idx];
        generalised_coordinates[3] = g->vrad[idx];
        generalised_coordinates[4] = g->vz[idx];
        generalised_coordinates[5] = g->vtor[idx];
        generalised_coordinates[BEAMLET_INTENSITY_ID] = g->intensity[idx];
        generalised_coordinates[TIME_OF_FLIGHT_ID] = g->time_of_flight[idx];
        
        g->detcellid[idx] = calculate_trajectory(c, generalised_coordinates, detcellid);
        
        g->rad[idx]  = generalised_coordinates[0];
        g->z[idx]    = generalised_coordinates[1];
        g->tor[idx]  = generalised_coordinates[2];
        g->vrad[idx] = generalised_coordinates[3];
        g->vz[idx]   = generalised_coordinates[4];
        g->vtor[idx] = generalised_coordinates[5];
        g->intensity[idx] = generalised_coordinates[BEAMLET_INTENSITY_ID];
        g->time_of_flight[idx] = generalised_coordinates[TIME_OF_FLIGHT_ID];
    }
    service_var[0] = 42.24;
}