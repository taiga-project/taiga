#include "hip/hip_runtime.h"
__global__ void taiga(TaigaGlobals *g, TaigaCommons *c, double *service_var){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;    // thread index
    
    if (g->detcellid[idx] == -1){
        double generalised_coordinates[6];
        int detcellid = g->detcellid[idx];
        
        generalised_coordinates[0] = g->rad[idx];
        generalised_coordinates[1] = g->z[idx];
        generalised_coordinates[2] = g->tor[idx];
        generalised_coordinates[3] = g->vrad[idx];
        generalised_coordinates[4] = g->vz[idx];
        generalised_coordinates[5] = g->vtor[idx];
        
        g->detcellid[idx] = calculate_trajectory(c, generalised_coordinates, detcellid);
        
        g->rad[idx]  = generalised_coordinates[0];
        g->z[idx]    = generalised_coordinates[1];
        g->tor[idx]  = generalised_coordinates[2];
        g->vrad[idx] = generalised_coordinates[3];
        g->vz[idx]   = generalised_coordinates[4];
        g->vtor[idx] = generalised_coordinates[5];
    }
    service_var[0] = 42.24;
}

__global__ void cuda_service_test(double *service_var){
    service_var[9] = 3.1415926535897932456;
}
