#include "hip/hip_runtime.h"
__device__ void copy_local_field(TaigaCommons *c,
                                 double position_rad, double position_z,
                                 int *local_spline_indices,
                                 double *local_spline_brad, double *local_spline_bz, double *local_spline_btor,
                                 double *local_spline_erad, double *local_spline_ez, double *local_spline_etor,
                                 double *local_polflux){
    int rci, zci;
    int i, i2;
    int rgrid_length = c->grid_size[0];
    int zgrid_length = c->grid_size[1];

    for(rci=0; (c->spline_rgrid[rci+1]<position_rad)&&(rci<rgrid_length-1); ++rci){;}

    for(zci=0; (c->spline_zgrid[zci+1]<position_z)&&(zci<zgrid_length-1); ++zci){;}

    // Particle leave out the cell
    if ((local_spline_indices[0] != rci) || (local_spline_indices[1] != zci)){
        local_spline_indices[0] = rci;
        local_spline_indices[1] = zci;

        for(i=0; i<16; ++i){
            i2 = (local_spline_indices[0])*(zgrid_length-1)+local_spline_indices[1];
            local_spline_brad[i] = c->brad[i][i2];
            local_spline_bz[i]   = c->bz[i][i2];
            local_spline_btor[i] = c->btor[i][i2];
            if (c->is_electric_field_on){
                local_spline_erad[i] = c->erad[i][i2];
                local_spline_ez[i]   = c->ez[i][i2];
                local_spline_etor[i] = c->etor[i][i2];
            }
            if (c->is_magnetic_field_perturbation){
                local_polflux[i] = c->polflux[i][i2];
            }
        }
    }
}

__device__ double calculate_local_field(double *local_spline, double dr, double dz){
    /* MATLAB CODE:
    sample2(3) =c11(bs1,bs2)*dsx^3*dsy^3 + c12(bs1,bs2)*dsx^3*dsy^2 + c13(bs1,bs2)*dsx^3*dsy + c14(bs1,bs2)*dsx^3 + ...
                c21(bs1,bs2)*dsx^2*dsy^3 + c22(bs1,bs2)*dsx^3*dsy^2 + c23(bs1,bs2)*dsx^2*dsy + c24(bs1,bs2)*dsx^2 + ...
                c31(bs1,bs2)*dsx  *dsy^3 + c32(bs1,bs2)*dsx  *dsy^2 + c33(bs1,bs2)*dsx  *dsy + c34(bs1,bs2)*dsx    + ...
                c41(bs1,bs2)      *dsy^3 + c42(bs1,bs2)      *dsy^2 + c43(bs1,bs2)      *dsy + c44(bs1,bs2);*/
    double local_field = 0.0, local_field_comp[16] ;
    for(int i=0; i<4; ++i){
        for(int j=0; j<4; ++j){
            local_field_comp[i*4+j] = local_spline[i*4+j]*pow(dr,3-i)*pow(dz,3-j);
        }
    }

    for(int i=0; i<4; ++i){
        for(int j=0; j<4; ++j){
            local_field += local_field_comp[i*4+j];
        }
    }
    return local_field;
}