#include "hip/hip_runtime.h"
#define SERVICE_VAR_LENGTH 10

#define ERRORCHECK() cErrorCheck(__FILE__, __LINE__)

#define HELP_MODE 1
#define HELP_DEVICES 2
#define HELP_VERSION 3

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <math.h>
#include <string.h>
//#include <filesystem>

#include <hip/hip_runtime_api.h>
//#include "test/cuda/nvToolsExt.h"

#include "utils/taiga_constants.h"
#include "utils/prop.h"
#include "main.cuh"
#include "interface/save.c"
#include "interface/feedback.c"
#include "utils/debug_functions.c"
#include "utils/basic_functions.h"

#include "dataio/data_import.c"
#include "dataio/field_import.cu"
#include "dataio/parameter_reader.c"

#include "init/beam.cu"
#include "init/init.c"
#include "init/sync.cu"
#include "init/detector.cu"
#include "init/fast_mode.cu"
#include "dataio/beam.h"
#if READINPUTPROF == 1
    #include "dataio/beam_manual_profile.c"
#elif RENATE == 110
    #include "dataio/beam_renate110.c"
#else
    #error A valid beam module is required!
#endif

#include "dataio/data_export.c"

#include "core/rk4.cu"
#include "core/solvers.cuh"
#include "core/verlet.cu"
#include "core/yoshida.cu"
#include "core/detection.cu"
#include "core/cyl2tor.cu"
#include "core/localise_field.cu"
#include "core/traj.cu"
#include "core/generate_coords.cu"
#include "core/taiga.cu"

#include "detector/module.cu"
#include "detector/postproc.cu"
#include "detector/sum.cu"

void input_init_taiga(int argc, char *argv[], ShotProp *shot, BeamProp *beam, RunProp *run){
    
    char *input;
    for (int i=1; i<argc; ++i){
        input = strtok(argv[i], "=");
        if (!strcmp(input, "--debug") || !strcmp(input, "-d")){
            run->debug = 1;
        }else if (!strcmp(input, "--fulltrace") || !strcmp(input, "-f")){
            run->step_host = 2000;
            run->step_device = 1;
        }else if (!strcmp(input, "--flux") || !strcmp(input, "-F")){
            run->magnetic_field_mode = MAGNETIC_FIELD_FROM_FLUX;
        }else if (!strcmp(input, "--help") || !strcmp(input, "-h")){
            run->help = HELP_MODE;
        }else if (!strcmp(input, "--devices") || !strcmp(input, "-D") || !strcmp(input, "-l")){
            run->help = HELP_DEVICES;
        }else if (!strcmp(input, "--parameter_file") || !strcmp(input, "-p")){
            input = strtok(NULL, "=");
            strcpy(run->parameter_file, input);
            printf("Parameter file: %s\n", run->parameter_file);
        }else if (!strcmp(input, "--runnumber_file")  || !strcmp(input, "-R")){
            strcpy(run->runnumber_file, input);
            printf("Runnumber file: %s\n", run->runnumber_file);
        }else if (!strcmp(input, "--runnumber") || !strcmp(input, "-r")){
            input = strtok(NULL, "=");
            strcpy(run->runnumber, input);
            strcpy(run->runnumber_file, "console init");
            printf("Runnumber: %s\n", run->runnumber);
        }else if (!strcmp(input, "--ion-source") || !strcmp(input, "-s")){
            input = strtok(NULL, "=");
            strcpy(run->ion_source_file, input);
            printf("Ion source file: %s\n", run->ion_source_file);
        }else if (!strcmp(input, "--ion-source-coords") || !strcmp(input, "-S")){
            input = strtok(NULL, "=");
            strcpy(run->io_coordinate_order, input);
            printf("Order of coordinates in input file: %s\n", run->io_coordinate_order);
        }else if (!strcmp(input, "--version") || !strcmp(input, "-v")){
            input = strtok(NULL, "=");
            run->help = HELP_VERSION;
        }else{
            printf("Warning: Undefined command line parameter: %s\n", input);
        }
    }
}

void print_help_message(){
    printf("%s\n", concat("TAIGA ", TAIGA_VERSION," (r", GIT_REV, ")", NULL));
    printf("Usage: taiga.exe [OPTION]\nOptions:\n");
    printf("  -d,      --debug                 Print additional debug informations\n");
    printf("  -D, -l,  --devices               List GPU devices\n");
    printf("  -f,      --fulltrace             Save coordinates at every timestep\n");
    printf("  -F,      --flux                  Import magnetic flux instead of magnetic field\n");
    printf("  -h,      --help                  Help message\n");
    printf("  -p=PATH, --parameter_file=PATH   Parameter file path\n");
    printf("  -r=INT,  --runnumber=INTEGER     Runnumber value\n");
    printf("  -R=PATH  --runnumber_file=PATH   Runnumber file path\n");
    printf("  -s=PATH, --ion-source=PATH       Ion source path\n");
    printf("  -S=XXX   --ion-source-coords=XXX Order of coordinates (RZT or RTZ) in input file\n");
    printf("  -v       --version               Version number\n");
}

void print_version(){
    printf("TAIGA (%s)\n\n", TAIGA_VERSION);
    printf("Trajectory simulator of ABP Ions with GPU Acceleration\n");
    printf("Copyright (C) 2011--2021\n\n");
    printf("Written by Matyas Aradi\n");
}

int main(int argc, char *argv[]){
    ShotProp shot; init_shot_prop(&shot);
    BeamProp beam; init_beam_prop(&beam);
    RunProp run;   init_run_prop(&run);
    input_init_taiga(argc, argv, &shot, &beam, &run);
    
    if (run.help == HELP_MODE){
        print_help_message();
    }else if (run.help == HELP_DEVICES){
        set_cuda(1);
    }else if (run.help == HELP_VERSION){
        print_version();
    }else{
        parameter_reader(&beam, &shot, &run);
        runnumber_reader(&shot, &run);
        
        init_dir(run.folder_out, run.runnumber);
        CopyFile(run.parameter_file, concat(run.folder_out,"/",run.runnumber,"/parameters.sh", NULL));
        
        //! CUDA profiler START
        hipProfilerStart();
        set_cuda(run.debug);
        
        TaigaGlobals *device_global, *host_global, *shared_global;
        TaigaCommons *device_common, *host_common, *shared_common;
        DetectorProp *shared_detector, *device_detector;
        
        size_t size_global = sizeof(TaigaGlobals);
        size_t size_commons = sizeof(TaigaCommons);
        size_t size_detector_prop = sizeof(DetectorProp);
        
        host_global = (TaigaGlobals*)malloc(size_global);
        shared_global = (TaigaGlobals*)malloc(size_global);
        host_common = (TaigaCommons*)malloc(size_commons);
        shared_common = (TaigaCommons*)malloc(size_commons);
        shared_detector = (DetectorProp*)malloc(size_detector_prop);
        
        hipMalloc((void **) &device_global, size_global);
        hipMalloc((void **) &device_common, size_commons);
        hipMalloc((void **) &device_detector, size_detector_prop);
        
        init_host(host_global, host_common);
        
        set_particle_number(&run, host_global, shared_global);
        
        //! coordinates
        init_coords(&beam, &shot, &run, host_global, shared_global);
        
        //! grid
        init_grid(shot, run, host_common, shared_common);
        magnetic_field_read_and_init(shot, run, host_common, shared_common);
        if (run.is_electric_field_on) run.is_electric_field_on = electric_field_read_and_init(shot, run, host_common, shared_common);
        if (run.is_magnetic_field_perturbation) run.is_magnetic_field_perturbation = poloidal_flux_read_and_init(shot, run, host_common, shared_common);

        // detector
        set_detector_geometry(shot, host_common, shared_common);
        init_detector(shared_detector, device_detector, shot);
        
        // <service value>
        size_t dimService = SERVICE_VAR_LENGTH * sizeof(double);
        double *host_service_array, *device_service_array;
        host_service_array = (double *)malloc(dimService);
        
        for(int i=0; i<SERVICE_VAR_LENGTH; ++i){
            host_service_array[i] = 0;
        }
        
        host_service_array[4] = 55555.55555;
        hipMalloc((void **) &device_service_array,  dimService);
        hipMemcpy(device_service_array, host_service_array, dimService, hipMemcpyHostToDevice);
        // </service value>
        
        if (!FASTMODE){
           save_trajectories(host_global, run);
        }
        
        print_run_details(host_global, host_common, shot, run);
        
        //! Set CUDA timer 
        hipEvent_t cuda_event_core_start, cuda_event_core_end, cuda_event_copy_start, cuda_event_copy_end;
        clock_t cpu_event_copy_start, cpu_event_copy_end;
        float cuda_event_core, cuda_event_copy;
        hipEventCreate(&cuda_event_core_start);
        hipEventCreate(&cuda_event_core_end);
        hipEventCreate(&cuda_event_copy_start);
        hipEventCreate(&cuda_event_copy_end);
        
        if (run.debug == 1 && !FASTMODE)   debug_message_init(host_global);
        
        size_t dimX = host_global->particle_number*sizeof(double);
        
        init_device_structs(beam, shot, run, shared_global, shared_common);
        sync_device_structs(device_global, shared_global, device_common, shared_common);
        if (FASTMODE)   init_fastmode(beam, shot, run, device_global);
        
        for (long step_i=0; step_i<run.step_host; ++step_i){
            if (step_i == 0) hipEventRecord(cuda_event_core_start, 0);
            
            taiga <<< run.block_number, run.block_size >>> (device_global, device_common, device_service_array);
            
            if (step_i == 0) hipEventRecord(cuda_event_core_end, 0);
            hipEventSynchronize(cuda_event_core_end);
            //ERRORCHECK();
            
            if (!FASTMODE){
                // ION COORDS (device2HOST)
                if (step_i == 0) hipEventRecord(cuda_event_copy_start, 0);
                coord_memcopy_back(beam, shot, run, host_global, shared_global);
                //ERRORCHECK();
                if (step_i == 0) hipEventRecord(cuda_event_copy_end, 0);
                
                // Save data to files
                cpu_event_copy_start = clock();
                save_trajectories(host_global, run);
                cpu_event_copy_end = clock();
            }
            
            if (run.debug == 1)    printf("Step\t%ld/%ld\n",step_i,run.step_host);
            if (run.debug == 1 && !FASTMODE)    debug_message_run(host_global);
        }
        
        // Get CUDA timer
        hipEventElapsedTime(&cuda_event_core, cuda_event_core_start, cuda_event_core_end);
        hipEventElapsedTime(&cuda_event_copy, cuda_event_copy_start, cuda_event_copy_end);
        if (!FASTMODE) run.cpu_time_copy = ((double) (4.0+run.step_host)*(cpu_event_copy_end - cpu_event_copy_start)) / CLOCKS_PER_SEC;
        run.cuda_time_copy = (double) (1.0+run.step_host)*cuda_event_copy/1000.0;
        run.cuda_time_core =  run.step_host*cuda_event_core/1000.0;
        printf("===============================\n");
        printf ("CUDA kernel runtime: %lf s\n", run.cuda_time_core);
        printf ("CUDA memcopy time:   %lf s\n", run.cuda_time_copy);
        if (!FASTMODE)  printf ("CPU->HDD copy time:  %lf s\n", run.cpu_time_copy);
        printf("===============================\n");
        
        //! MEMCOPY (device2HOST)
        hipMemcpy(host_service_array, device_service_array, dimService, hipMemcpyDeviceToHost);
        if(host_service_array[0] != 42.24){
            printf("\n +----------------------------+\n | Fatal error in running.    | \n | The CUDA did not run well. |\n | Service value: %11lf |\n +----------------------------+\n\n", host_service_array[0]);
        }else{
            printf("\nSuccessful run. \n\n");
        }
        
        detector_postproc <<< run.block_number, run.block_size >>> (device_global, device_common, device_detector);
        detector_sum <<<1,1>>> (device_global, device_common, device_detector);
        export_detector(shared_detector, device_detector, shared_global, shot, run);
        
        //! CUDA profiler STOP
        hipProfilerStop();
        
        if (run.debug == 1)    debug_service_vars(host_service_array);
        
        fill_header_file(host_common, beam, shot, run);
        
        if (!FASTMODE){
            save_endpoints(host_global, run);
        }
        
        printf("\nData folder: %s/%s\n\n", run.folder_out, run.runnumber);
        
        //! FREE host_service_array variables (RAM, cuda)
        free(host_service_array);  hipFree(device_service_array);
        printf("Ready.\n\n");
    }
}
