#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "utils/taiga_constants.h"
#include "utils/prop.h"
#include "utils/basic_functions.h"
#include "dataio/data_import.c"
#include "dataio/field_import.cu"
#include "dataio/parameter_reader.c"
#include "init/sync.cu"
#include "init/init.c"
#include "core/cyl2tor.cu"
#include "core/detection.cu"
#include "core/localise_field.cu"

#define GRID_RES 101

__global__ void calculate_field_grid(TaigaCommons *c, double *R, double *Z, double *field, double *polflux){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double r = R[idx];
    double z = Z[idx];
    
    int local_spline_indices[2];
    local_spline_indices[0] = SPLINE_INDEX_ERROR;
    local_spline_indices[1] = SPLINE_INDEX_ERROR;

    double local_spline_brad[16];
    double local_spline_bz[16];
    double local_spline_btor[16];

    double local_spline_erad[16];
    double local_spline_ez[16];
    double local_spline_etor[16];

    double local_spline_polflux[16];

    double local_brad=0, local_bz=0, local_btor=0;
    double local_erad=0, local_ez=0, local_etor=0;
    double local_polflux = 0;
    double dr, dz;

    copy_local_field(c, r, z, local_spline_indices,
                     local_spline_brad, local_spline_bz, local_spline_btor,
                     local_spline_erad, local_spline_ez, local_spline_etor,
                     local_spline_polflux);
                 
    dr = r-c->spline_rgrid[local_spline_indices[0]];
    dz = z-c->spline_zgrid[local_spline_indices[1]];
    local_brad = calculate_local_field_with_splines(c, local_spline_indices, local_spline_brad, dr, dz);
    local_bz   = calculate_local_field_with_splines(c, local_spline_indices, local_spline_bz,   dr, dz);
    local_btor = calculate_local_field_with_splines(c, local_spline_indices, local_spline_btor, dr, dz);
    local_polflux = calculate_local_field_with_splines(c, local_spline_indices, local_spline_polflux, dr, dz);
    field[idx] = local_brad;
    field[idx+GRID_RES*GRID_RES] = local_bz;
    field[idx+2*GRID_RES*GRID_RES] = local_btor;
    polflux[idx] = local_polflux;
}

void test_field(int field_interpolation_method){
    ShotProp shot; init_shot_prop(&shot);
    BeamProp beam; init_beam_prop(&beam);
    RunProp run;   init_run_prop(&run);
    
    parameter_reader(&beam, &shot, &run);
    runnumber_reader(&shot, &run);
    
    TaigaGlobals *device_global, *host_global, *shared_global;
    TaigaCommons *device_common, *host_common, *shared_common;
    
    size_t size_global = sizeof(TaigaGlobals);
    size_t size_commons = sizeof(TaigaCommons);
    
    host_global = (TaigaGlobals*)malloc(size_global);
    shared_global = (TaigaGlobals*)malloc(size_global);
    host_common = (TaigaCommons*)malloc(size_commons);
    shared_common = (TaigaCommons*)malloc(size_commons);
    
    hipMalloc((void **) &device_global, size_global);
    hipMalloc((void **) &device_common, size_commons);
    
    init_host(host_global, host_common);
    run.field_interpolation_method = field_interpolation_method;
    init_grid(shot, run, host_common, shared_common);
    magnetic_field_read_and_init(shot, run, host_common, shared_common);

    run.is_magnetic_field_perturbation = true;
    poloidal_flux_read_and_init(shot, run, host_common, shared_common);
    
    init_device_structs(beam, shot, run, shared_global, shared_common);
    sync_device_structs(device_global, shared_global, device_common, shared_common);
    
    double *host_field, *device_field;
    double *host_R, *device_R;
    double *host_Z, *device_Z;
    double *host_polflux, *device_polflux;
    long grid_size = GRID_RES*GRID_RES;
    size_t dim_tmp = sizeof(double)*grid_size;
    host_field = (double *) malloc(3*dim_tmp);
    host_R = (double *) malloc(dim_tmp);
    host_Z = (double *) malloc(dim_tmp);
    host_polflux = (double *) malloc(dim_tmp);
    hipMalloc((void **) &(device_field), 3*dim_tmp);
    hipMalloc((void **) &(device_R), dim_tmp);
    hipMalloc((void **) &(device_Z), dim_tmp);
    hipMalloc((void **) &(device_polflux), dim_tmp);
    
    double R_max=0.8;
    double R_min=0.3;
    double Z_max=0.4;
    double Z_min=-0.4;
    
    for(int i=0; i<GRID_RES; ++i){
        for(int j=0; j<GRID_RES; ++j){
            int index=i*GRID_RES+j;
            host_R[index] = i*(R_max-R_min)/GRID_RES+R_min;
            host_Z[index] = j*(Z_max-Z_min)/GRID_RES+Z_min;
            host_field[index]=UNDEFINED_FLOAT;
            host_field[grid_size+index]=UNDEFINED_FLOAT;
            host_field[2*grid_size+index]=UNDEFINED_FLOAT;
            host_polflux[index]=UNDEFINED_FLOAT;
        }
    }
    
    hipMemcpy(device_field, host_field, 3*dim_tmp, hipMemcpyHostToDevice);
    hipMemcpy(device_R, host_R, dim_tmp, hipMemcpyHostToDevice);
    hipMemcpy(device_Z, host_Z, dim_tmp, hipMemcpyHostToDevice);
    hipMemcpy(device_polflux, host_polflux, dim_tmp, hipMemcpyHostToDevice);
    
    calculate_field_grid <<< GRID_RES, GRID_RES >>> (device_common, device_R, device_Z, device_field, device_polflux);
    
    hipMemcpy(host_field, device_field, 3*dim_tmp, hipMemcpyDeviceToHost);
    hipMemcpy(host_polflux, device_polflux, dim_tmp, hipMemcpyDeviceToHost);
    
    FILE *fp;
    fp = fopen ("exported_fieldR.dat", "w");
    for (int i=0; i<grid_size; ++i){
        fprintf(fp, "%lf %lf %lf\n", host_R[i], host_Z[i], host_field[i]);
    }
    fclose(fp);
    
    fp = fopen ("exported_fieldZ.dat", "w");
    for (int i=0; i<grid_size; ++i){
        fprintf(fp, "%lf %lf %lf\n", host_R[i], host_Z[i], host_field[grid_size+i]);
    }
    fclose(fp);
    
    fp = fopen ("exported_fieldT.dat", "w");
    for (int i=0; i<grid_size; ++i){
        fprintf(fp, "%lf %lf %lf\n", host_R[i], host_Z[i], host_field[2*grid_size+i]);
    }
    fclose(fp);

    fp = fopen ("exported_polflux.dat", "w");
    for (int i=0; i<grid_size; ++i){
        fprintf(fp, "%lf %lf %lf\n", host_R[i], host_Z[i], host_polflux[i]);
    }
    fclose(fp);
}

int main(){
    test_field(CUBIC_SPLINE);
    //test_field(CUBIC_BSPLINE);
}