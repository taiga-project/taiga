#include "hip/hip_runtime.h"
// TAIGA default parameters

#define $R_defl 2.3                 //! radial position of deflection plates in meter -> TOROIDAL DEFLECTION

#define PI 3.141592653589792346
#define ELEMENTARY_CHARGE 1.60217656535e-19
#define AMU 1.66053892173e-27
#define INFINITY RAND_MAX

#define ERRORCHECK() cErrorCheck(__FILE__, __LINE__)

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#include <string.h>
//#include <filesystem>

#include <hip/hip_runtime_api.h>
#include "test/cuda/nvToolsExt.h"

#include "prop.h"
#include "main.cuh"
#include "debug_functions.c"
#include "basic_functions.c"
#include "dataio/data_import.c"
#include "dataio/field_import.c"
#include "dataio/parameter_reader.c"

#if READINPUTPROF == 1
    #include "dataio/beam_manual_profile.c"
#elif RENATE == 110
    #include "dataio/beam_renate110.c"
#else
    #error A valid beam module is required!
#endif

#include "dataio/data_export.c"

#include "running/rk4.cu"
#include "running/detection.cu"
#include "running/undetected.cu"
#include "running/cyl2tor.cu"
#include "running/traj.cu"
#include "running/taiga.cu"

#include "detector_module.c"
//#include "running/detector_postproc.cu"

void input_init_taiga(int argc, char *argv[], shot_prop *shot, beam_prop *beam, run_prop *run){
    
    char *input;
    for (int i=1; i<argc; i++){
        input = strtok(argv[i], "=");
        if (!strcmp(input, "--debug") || !strcmp(input, "-d")){
            run->debug = 1;
        }else if (!strcmp(input, "--fulltrace") || !strcmp(input, "-f")){
            run->step_host = 2000;
            run->step_device = 1;
        }else if (!strcmp(input, "--help") || !strcmp(input, "-h")){
            run->help = 1;
        }else if (!strcmp(input, "--devices") || !strcmp(input, "-l")){   
            run->help = 2;
        }else if (!strcmp(input, "--parameter_file") || !strcmp(input, "-p")){
            input = strtok(NULL, "=");
            strcpy(run->parameter_file, input);
            printf("Parameter file: %s\n", run->parameter_file);
        }else if (!strcmp(input, "--runnumber_file") || !strcmp(input, "--runnumber") || !strcmp(input, "-r")){
            input = strtok(NULL, "=");
            int runnumber = atoi(input);
            if (runnumber || !strcmp(input, "0")){
                run->runnumber = runnumber;
                strcpy(run->runnumber_file, "console init");
                printf("Runnumber: %d\n", run->runnumber);            
            }else{
                strcpy(run->runnumber_file, input);
                printf("Runnumber file: %s\n", run->runnumber_file);
            }
        }else if (!strcmp(input, "--ion-source") || !strcmp(input, "-s")){
            input = strtok(NULL, "=");
            strcpy(run->ion_source_file, input);
            printf("Ion source file: %s\n", run->ion_source_file);
        }else if (!strcmp(input, "--ion-source-coords")){
            input = strtok(NULL, "=");
            strcpy(run->io_coordinate_order, input);
            printf("Order of coordinates in input fil: %s\n", run->io_coordinate_order);
        }
    }
}

void print_help_message(){        
        printf("%s\n", concat("TAIGA ", TAIGA_VERSION," (r", SVN_REV, ")"));
        printf("Usage: taiga.exe [options]\nOptions:\n");
        printf("  -d, --debug                 Print additional debug informations\n");
        printf("  -f, --fulltrace             Save coordinates at every timestep\n");
        printf("  -h, --help                  Help message\n");
        printf("  -l, --devices               List GPU devices\n");
        printf("  -p, --parameter_file=PATH   Parameter file path\n");
        printf("  -r  --runnumber_file=PATH   Runnumber file path\n");
        printf("  -r  --runnumber=INTEGER     Runnumber value\n");
        printf("  -s, --ion-source=PATH       Ion source path\n");
        printf("  -s, --ion-source=PATH       Ion source path\n");
        printf("      --ion-source-coords=XXX Order of coordinates (RZT or RTZ) in input file\n");
}

int main(int argc, char *argv[]){    
    shot_prop shot;
    beam_prop beam;
    run_prop run;
    input_init_taiga(argc, argv, &shot, &beam, &run);
    
    if (run.help == 1){
        print_help_message();
    }else if (run.help == 2){
        set_cuda(1);
    }else{  
        parameter_reader(&shot, &beam, &run);
        runnumber_reader(&shot, &run);
        
        char* folder_out=concat("results/", shot.name);        
        char timestamp[80];
        sprintf(timestamp, "%d", run.runnumber);
        
        init_dir(folder_out, timestamp);
        CopyFile(run.parameter_file, concat(folder_out,"/",timestamp,"/parameters.sh"));

        size_t dimD = 5 * sizeof(double);
        double *DETECTOR, *detector;
        DETECTOR = (double *)malloc(dimD);  hipMalloc((void **) &detector,  dimD); 
        
        set_detector_geometry(DETECTOR, shot.detector_geometry);
        
        printf("%s\n", concat("TAIGA ", TAIGA_VERSION," (r", SVN_REV, ")"));
        printf("Shotname: %s\n", shot.name); 
        printf("Detector: %s\n", shot.detector_mask);
        printf("  R:\t%lf\n", DETECTOR[0]);
        printf("  Z:\t%lf\n", DETECTOR[1]);
        printf("  T:\t%lf\n", DETECTOR[2]);
        printf("  angle (Z/R):\t%lf°\n", atan(DETECTOR[3])/PI*180.0);
        printf("  angle (T/R):\t%lf°\n", atan(DETECTOR[4])/PI*180.0);
        printf("===============================\n");

        int NX = run.block_size * run.block_number;

        if (READINPUTPROF == 1){
            double *XR;
            NX = read_vector(&XR, "input", "manual_profile", "rad.dat");
            run.block_number = NX / run.block_size+1;
        }
        
        set_cuda(run.debug);

        // set timestamp
        time_t rawtime;
        struct tm *info;

        // coords
        double *X_PTR[3], **x_ptr;
        double *V_PTR[3], **v_ptr;
        size_t dimXP = 3*sizeof(double*);

        double *XR,  *xr; 
        double *XZ,  *xz;
        double *XT,  *xt;

        double *VR,  *vr; 
        double *VZ,  *vz;
        double *VT,  *vt;

        printf("Number of blocks (threads): %d\n", run.block_number);
        printf("Block size: %d\n", run.block_size);
        printf("Number of particles: %d\n", NX);
        printf("Max steps on device (GPU): %d\n", run.step_device);
        printf("Max steps on host (HDD): %d\n", run.step_host);


        //! position and velocity array allocation
        size_t dimX = run.block_size * run.block_number * sizeof(double);
        
        XR = (double*)malloc(dimX);
        XZ = (double*)malloc(dimX);
        XT = (double*)malloc(dimX);

        VR = (double*)malloc(dimX);
        VZ = (double*)malloc(dimX);
        VT = (double*)malloc(dimX);

        // phys. constants
        double eperm = ELEMENTARY_CHARGE/ AMU/ beam.mass;

        load_beam(XR, XZ, XT, VR, VZ, VT, beam, shot, run);

        hipMalloc((void **) &xr,  dimX); 
        hipMalloc((void **) &xz,  dimX); 
        hipMalloc((void **) &xt,  dimX); 
        hipMalloc((void **) &x_ptr,  dimXP); 

        hipMalloc((void **) &vr,  dimX); 
        hipMalloc((void **) &vz,  dimX); 
        hipMalloc((void **) &vt,  dimX); 
        hipMalloc((void **) &v_ptr,  dimXP); 

        //! coords pointers
        X_PTR[0] = xr;
        X_PTR[1] = xz;
        X_PTR[2] = xt;

        V_PTR[0] = vr;
        V_PTR[1] = vz;
        V_PTR[2] = vt;
        
        //! grid pointers
        double *G_PTR[2];
        double **g_ptr;
        size_t dimG = 2*sizeof(double*);
        hipMalloc((void **) &g_ptr,  dimG); 
        double *RG, *rg;
        double *ZG, *zg;

        // size definitions

        //! R-grid points
        int NR = read_vector(&RG, "input/fieldSpl", shot.name, "r.spline");
        size_t dimR = NR * sizeof(double);
        hipMalloc((void **) &rg,  dimR); 
        
        //! Z-grid points
        int NZ = read_vector(&ZG, "input/fieldSpl", shot.name, "z.spline");
        size_t dimZ = NZ * sizeof(double);
        size_t dimRZ = (NR-1) * (NZ-1) * sizeof(double);
        hipMalloc((void **) &zg,  dimZ); 

        // grid pointer
        G_PTR[0] = rg;
        G_PTR[1] = zg;

        //! MAGN. FIELD (HOST, device) ALLOCATION  
        double **br_ptr, **bz_ptr, **bt_ptr;
        double **er_ptr, **ez_ptr, **et_ptr;
        
        int magnetic_field_loaded = magnetic_field_read_and_init(shot, run, &br_ptr,&bz_ptr,&bt_ptr, dimRZ);
        if (shot.electric_field_module) shot.electric_field_module = electric_field_read_and_init(shot, run, &er_ptr,&ez_ptr,&et_ptr, dimRZ);
        
        // detector cell id
        size_t dimRint = NX * sizeof(int);
        int *DETCELLID, *detcellid;
        DETCELLID = (int *)malloc(dimRint); hipMalloc((void **) &detcellid,  dimRint);
        
        // service value
        size_t dimService = 10 * sizeof(double);
        double *SERVICE_VAR, *service_var;
        SERVICE_VAR = (double *)malloc(dimService); hipMalloc((void **) &service_var,  dimService);

        //! CUDA profiler START
        hipProfilerStart();
        
        //! MEMCOPY (HOST2device)

        //! GRID COORDS
        hipMemcpy(rg, RG, dimR, hipMemcpyHostToDevice);
        hipMemcpy(zg, ZG, dimZ, hipMemcpyHostToDevice);
        hipMemcpy(g_ptr, G_PTR, dimG, hipMemcpyHostToDevice);

        //! ION COORDS (HOST2device)
        hipMemcpy(x_ptr, X_PTR, dimXP, hipMemcpyHostToDevice);

        //! ION SPEEDS (HOST2device)
        hipMemcpy(v_ptr, V_PTR, dimXP, hipMemcpyHostToDevice);

        //! DETECTOR COORDS (HOST2device)
        hipMemcpy(detector, DETECTOR, dimD, hipMemcpyHostToDevice);
        
        // OUTPUT INIT
        export_data(XR, NX, folder_out, timestamp, "t_rad.dat");
        export_data(XZ, NX, folder_out, timestamp, "t_z.dat");
        export_data(XT, NX, folder_out, timestamp, "t_tor.dat");
        export_data(VR, NX, folder_out, timestamp, "t_vrad.dat");
        export_data(VZ, NX, folder_out, timestamp, "t_vz.dat");
        export_data(VT, NX, folder_out, timestamp, "t_vtor.dat");

        //! Set CUDA timer 
        hipEvent_t cuda_event_core_start, cuda_event_core_end, cuda_event_copy_start, cuda_event_copy_end;
        clock_t cpu_event_copy_start, cpu_event_copy_end;
        double cpu_time_copy, cuda_time_core, cuda_time_copy;
        float cuda_event_core, cuda_event_copy;
        hipEventCreate(&cuda_event_core_start);
        hipEventCreate(&cuda_event_core_end);
        hipEventCreate(&cuda_event_copy_start);
        hipEventCreate(&cuda_event_copy_end);

        if (run.debug == 1)    debug_message_init(XR, XZ, XT, VR, VZ, VT);
        
        for (int step_i=0;step_i<run.step_host;step_i++){        
            
            if (step_i == 0) hipEventRecord(cuda_event_copy_start, 0);
            // ION COORDS (HOST2device)
            hipMemcpy(xr, XR, dimX, hipMemcpyHostToDevice);
            hipMemcpy(xz, XZ, dimX, hipMemcpyHostToDevice);
            hipMemcpy(xt, XT, dimX, hipMemcpyHostToDevice);
            //hipMemcpy(x_ptr, X_PTR, dimXP, hipMemcpyHostToDevice);

            // ION SPEEDS (HOST2device)
            hipMemcpy(vr, VR, dimX, hipMemcpyHostToDevice);
            hipMemcpy(vz, VZ, dimX, hipMemcpyHostToDevice);
            hipMemcpy(vt, VT, dimX, hipMemcpyHostToDevice);
            //hipMemcpy(v_ptr, V_PTR, dimXP, hipMemcpyHostToDevice);   
            //ERRORCHECK();
            
            if (step_i == 0) hipEventRecord(cuda_event_copy_end, 0);
            if (step_i == 0) hipEventRecord(cuda_event_core_start, 0);
            
            if (shot.electric_field_module){
                printf("electric_field_module ON\n");
                taiga <<< run.block_number, run.block_size >>> (run.timestep,NR,NZ,eperm,br_ptr,bz_ptr,bt_ptr,er_ptr,ez_ptr,et_ptr,g_ptr,x_ptr,v_ptr,detector,detcellid,run.step_device,service_var,step_i);
            }else{
                taiga <<< run.block_number, run.block_size >>> (run.timestep,NR,NZ,eperm,br_ptr,bz_ptr,bt_ptr,g_ptr,x_ptr,v_ptr,detector,detcellid,run.step_device,service_var,step_i);
            }
            if (step_i == 0) hipEventRecord(cuda_event_core_end, 0);
            hipEventSynchronize(cuda_event_core_end);
            ERRORCHECK();

            // ION COORDS (device2HOST)
            hipMemcpy(XR, xr, dimX, hipMemcpyDeviceToHost);
            hipMemcpy(XZ, xz, dimX, hipMemcpyDeviceToHost);
            hipMemcpy(XT, xt, dimX, hipMemcpyDeviceToHost);
            //ERRORCHECK();
            
            // ION SPEEDS (device2HOST)
            hipMemcpy(VR, vr, dimX, hipMemcpyDeviceToHost);
            hipMemcpy(VZ, vz, dimX, hipMemcpyDeviceToHost);
            hipMemcpy(VT, vt, dimX, hipMemcpyDeviceToHost);
            //ERRORCHECK();
            
            // Save data to files
            cpu_event_copy_start = clock();  
            export_data(XR, NX, folder_out, timestamp, "t_rad.dat");
            export_data(XZ, NX, folder_out, timestamp, "t_z.dat");
            export_data(XT, NX, folder_out, timestamp, "t_tor.dat");
            export_data(VR, NX, folder_out, timestamp, "t_vrad.dat");
            export_data(VZ, NX, folder_out, timestamp, "t_vz.dat");
            export_data(VT, NX, folder_out, timestamp, "t_vtor.dat");
            cpu_event_copy_end = clock();
            
            if (run.debug == 1)    printf("Step\t%d/%d\n",step_i,run.step_host);
            if (run.debug == 1)    debug_message_run(XR, XZ, XT, VR, VZ, VT);
        }

        // Get CUDA timer 
        hipEventElapsedTime(&cuda_event_core, cuda_event_core_start, cuda_event_core_end);
        hipEventElapsedTime(&cuda_event_copy, cuda_event_copy_start, cuda_event_copy_end);
        cpu_time_copy = ((double) (4.0+run.step_host)*(cpu_event_copy_end - cpu_event_copy_start)) / CLOCKS_PER_SEC;
        cuda_time_copy = (double) 2.0*run.step_host*cuda_event_copy/1000.0;
        cuda_time_core =  run.step_host*cuda_event_core/1000.0;
        
        printf("===============================\n");
        printf ("CUDA kernel runtime: %lf s\n", cuda_time_core);
        printf ("CUDA memcopy time:   %lf s\n", cuda_time_copy);
        printf ("CPU->HDD copy time:  %lf s\n", cpu_time_copy);    
        printf("===============================\n");
        
        undetected <<<1,1>>>(detcellid, NX, service_var);

        //! MEMCOPY (device2HOST)
        hipMemcpy(SERVICE_VAR, service_var, dimService, hipMemcpyDeviceToHost);
        if(SERVICE_VAR[0]!=42.24){
            printf("\n +----------------------------+\n | Fatal error in running.    | \n | The CUDA did not run well. |\n | Service value: %11lf |\n +----------------------------+\n\n", SERVICE_VAR[0]);
        }else{
            printf("\nSuccessful run. \n\n");
        }

        printf("Lost particle ratio: \t %.4lf % \n", SERVICE_VAR[1]*100);
        
        detector_module(x_ptr, detector, detcellid, shot.detector_mask, run.block_number, run.block_size, NX, folder_out, timestamp);
        hipMemcpy(DETCELLID, detcellid, dimRint, hipMemcpyDeviceToHost);
        export_data(DETCELLID, NX, folder_out, timestamp, "detector", "cellid.dat");

        //! CUDA profiler STOP
        hipProfilerStop();

        export_header(concat("TAIGA ", TAIGA_VERSION," (r", SVN_REV, ")"), folder_out, timestamp);
        export_header_addline(folder_out, timestamp);
        export_header(concat("Shot ID: ",shot.name), folder_out, timestamp);
        export_header(concat("Run ID:  ",timestamp), folder_out, timestamp);
        export_header_addline(folder_out, timestamp);
        export_header("ABP ION TRAJECTORIES", folder_out, timestamp);

        if(READINPUTPROF==1){
            export_header("Manual (6D) input profile", folder_out, timestamp);
        }else if(RENATE==110){
            export_header("TS + Renate 1.1.0 input profile", folder_out, timestamp);
        }
        export_header_addline(folder_out, timestamp);

        if(!READINPUTPROF){
            export_header("Beam energy", "keV", beam.energy, folder_out, timestamp);
            export_header("Atomic mass", "AMU", beam.mass, folder_out, timestamp);
            export_header("Beam diameter", "mm", beam.diameter*1000, folder_out, timestamp);
            export_header("Beam deflection (toroidal/vertical)", "°", beam.toroidal_deflection*180.0/PI, beam.vertical_deflection*180.0/PI, folder_out, timestamp);
        }
        
        export_header("Number of ions", "", (double)NX, folder_out, timestamp);
        export_header_addline(folder_out, timestamp);
        export_header("Detector position (R)", "m", DETECTOR[0], folder_out, timestamp);
        export_header("Detector position (Z)", "m", DETECTOR[1], folder_out, timestamp);
        export_header("Detector position (T)", "m", DETECTOR[2], folder_out, timestamp);
        export_header("Detector angle (Z/R)", "°", atan(DETECTOR[3])*180.0/PI, folder_out, timestamp);
        export_header("Detector angle (T/R)", "°", atan(DETECTOR[4])*180.0/PI, folder_out, timestamp);
        export_header(concat("Detector mask:  \t", shot.detector_mask), folder_out, timestamp);
        export_header_addline(folder_out, timestamp);
        export_header("Timestep", "s", run.timestep, folder_out, timestamp);
        export_header_addline(folder_out, timestamp);
        export_header("CUDA kernel runtime", "s", cuda_time_core, folder_out, timestamp);
        export_header("CUDA memcopy time", "s", cuda_time_copy, folder_out, timestamp);
        export_header("CPU->HDD copy time", "s", cpu_time_copy, folder_out, timestamp);
        export_header_addline(folder_out, timestamp);
        export_header("Number of blocks (threads)", "", run.block_number, folder_out, timestamp);
        export_header("Block size", "", run.block_size, folder_out, timestamp);
        export_header("Length of a loop", "", run.step_device, folder_out, timestamp);
        export_header("Number of loops", "", run.step_host, folder_out, timestamp);

        //! Save data to files
        export_data(XR, NX, folder_out, timestamp, "rad.dat");
        export_data(XZ, NX, folder_out, timestamp, "z.dat");
        export_data(XT, NX, folder_out, timestamp, "tor.dat");
        export_data(VR, NX, folder_out, timestamp, "vrad.dat");
        export_data(VZ, NX, folder_out, timestamp, "vz.dat");
        export_data(VT, NX, folder_out, timestamp, "vtor.dat");
        export_table(folder_out, timestamp, "coords.dat", NX, XR, "R [m]", XZ, "Z [m]", XT, "T [m]", VR, "v_R [m/s]", VZ, "v_Z [m/s]", VT, "v_T [m/s]");

        printf("\n\nData folder: %s/%s\n\n", folder_out, timestamp);

        //! Free CUDA
        hipFree(x_ptr);    hipFree(xr);   hipFree(xz);   hipFree(xt);
        hipFree(g_ptr);    hipFree(rg);   hipFree(zg);
        hipFree(br_ptr);   hipFree(bz_ptr);   hipFree(bt_ptr);
        hipFree(er_ptr);   hipFree(ez_ptr);   hipFree(et_ptr);

        //! Free RAM
        free(RG);   free(ZG);
        free(XR);   free(XZ);   free(XT);
        
        //! FREE SERVICE_VAR variables (RAM, cuda)
        free(SERVICE_VAR);  hipFree(service_var);

        printf("Ready.\n\n");
    }
}
