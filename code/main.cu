#include "hip/hip_runtime.h"
// TAIGA default parameters

#define BANANA		 0		//! @param BANANA ABP: 0, banana orbits: 1
 
#define RADIONS		 1		//! @param RADIONS Real ion positions: 1, R=const 0

#define $ELM		 0		//! @param $ELM turn on <<ELM current perturbation>> mode

#define RKOLD		 0		//! @param RKOLD do not set! 0 (semi-RK: 1)

#define $3DINPUTPROF 1

#define $RENATE		0//110

#define N_BLOCKS     192		//! @param N_BLOCKS Number of blocks (max 192 on Geforce GTS450) (max 768 on Geforce GTS650Ti)
#define BLOCK_SIZE 	 1//30*4 		//! @param BLOCK_SIZE smaller is better (max 1M)

#define R_midions	 0.695		//! @param R_midions mid of ions at BANANA and no-RADIONS

#define $R_defl		2.3			//! radial position of deflection plates in meter -> TOROIDAL DEFLECTION

#define $deflH	 0				//! @param $deflH horizontal deflection in rad (up--down)  
#define $deflV	 0				//! @param $deflV vertical deflection in rad (left--right) -> TOROIDAL DEFLECTION


#if BANANA == 1
    #define $energy   0.5            // in keV
    #define $mass     2.013553212724 // in AMU (D)
    #define $diameter 50e-20         // in mm 
    #define $DETPOS 1 //! detector position
	#define dt		 1e-12			// timestep in seconds
	#define Nstep	 100000//00		// max step of a loop
	#define Nloop	 1000			// number of loops	
#else
	#define $energy   60				//! @param energy in keV
	#define $mass     7.016004558	//! @param mass in AMU (Li-7)
    #define $DETPOS 0.7089 //! detector position
    #define $diameter 25//4/*e-20*/      //! @param diameter in mm
    #define dt       1e-9			//! @param dt timestep in seconds
    #define Nstep    2000//000			//! @param Nstep max step of a loop
    #define Nloop    1//000				//! @param Nloop number of loops

#endif



#define ERRORCHECK() cErrorCheck(__FILE__, __LINE__)
#define PI 3.141592653589792346

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#include <string.h>

#include <hip/hip_runtime_api.h>
#include "cuda/nvToolsExt.h"

#include "dataio/fieldIn.c"

#if BANANA == 1
	#include "dataio/beamInBan.c"
#elif RADIONS == 1
	#if $3DINPUTPROF == 1
		#include "dataio/beamInFull.c"
	#elif $RENATE == 110
		#include "dataio/beamInRenate110.c"
	#else
		#include "dataio/beamIn.c"
	#endif
#else
	#include "dataio/beamInOne.c"
#endif
#include "dataio/beamOut.c"




#if RKOLD == 0
	#include "running/rk4.cu"
#else
	#include "running/rk4old.cu"
#endif

#include "running/ipol.cu"
#include "running/cyl2tor.cu"


#if RKOLD == 0
	#include "running/traj.cu"
#else
	#include "running/trajold.cu"
#endif


#include "running/ctrl.cu"

char* concat(const char *s1, const char *s2);


struct beam_prop{
    char* matter = "Li";
    double mass = 7.016004558;
    double energy = (double)$energy ;
    double diameter = (double)$diameter;
    double toroidal_deflation = (double)$deflH;   
    double vertical_deflation = (double)$deflV;
    double detector_R = (double)$DETPOS;
    
};

struct shot_prop{
    char* name = "11347";
    int runnumber = 0;  
    int electric_field_module = 0;
    int debug = 0;
    int block_size = BLOCK_SIZE;
    int block_number = N_BLOCKS;
    int step_host = 1; // on HDD
    int step_device = 2000; // on GPU
};

inline void cErrorCheck(const char *file, int line) {
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(err));
    printf(" @ %s: %d\n", file, line);
    exit(-1);
  }
}

int set_cuda(){
	int num_devices, device, max_device;
	hipGetDeviceCount(&num_devices);
	printf("Number of devices: %d\n",num_devices);
	
	if (num_devices > 1) {
        int max_multiprocessors = 0, max_device = 0;
        for (device = 0; device < num_devices; device++) {
            hipDeviceProp_t properties;
            hipGetDeviceProperties(&properties, device);
            if (max_multiprocessors < properties.multiProcessorCount) {
                max_multiprocessors = properties.multiProcessorCount;
                max_device = device;
            }
	      /*  printf("%d:%s\n",device,&properties.name);
	        printf("\tL2Cache:\t%d",	properties.l2CacheSize);
	        printf("\tNumber of cores:\t%d",	properties.warpSize);
	
	        printf("\tKernels:\t%d",	properties.concurrentKernels);
	        printf("\tThreads:\t%d",	properties.maxThreadsPerMultiProcessor);
	        printf("\tClock:\t%d",	properties.clockRate/1024);
	        printf("\n");*/
        }
        hipSetDevice(max_device);
        for (device = 0; device < num_devices; device++) {
        	if(device==max_device) printf("-->");
            hipDeviceProp_t properties;
            hipGetDeviceProperties(&properties, device);
        	printf("\t%d:\t%s\n",device,&properties.name);
        }
        
    }
	
	hipDeviceProp_t prop;
	hipGetDevice(&max_device);
	hipGetDeviceProperties(&prop, 0) ;  
}

double get_mass(char *s){
    double mass;
    
    if (strcmp(s,"D")==0){
        mass = 2.013553212724;
    }else if (strcmp(s,"Li")==0){
        mass = 7.016004558;
    }else if (strcmp(s,"Na")==0){
        mass = 20.0073517;
    }else if (strcmp(s,"K")==0){
        mass = 39.9639984821;
    }else if (strcmp(s,"H2")==0){
        mass = 2.013553212724;
    }else if (strcmp(s,"Li7")==0){
        mass = 7.016004558;
    }else if (strcmp(s,"Na20")==0){
        mass = 20.0073517;
    }else if (strcmp(s,"K40")==0){
        mass = 39.9639984821;
    }else{
        mass = (double)$mass;
    }
    
    return mass;

}



int spline_read_and_init(shot_prop shot, char* field_name, double ***return_s_ptr, int dimRZ){

	char* spline_folder = "input/fieldSpl";
	int suc[1] = {1};
    
	double *S0,  *s0;  vectorReader(&S0, "input/fieldSpl", shot.name, concat(field_name ,".spl11"), suc);	hipMalloc((void **) &s0,  dimRZ); 
	double *S1,  *s1;  vectorReader(&S1, "input/fieldSpl", shot.name, concat(field_name ,".spl12"), suc);	hipMalloc((void **) &s1,  dimRZ);
	double *S2,  *s2;  vectorReader(&S2, "input/fieldSpl", shot.name, concat(field_name ,".spl13"), suc);	hipMalloc((void **) &s2,  dimRZ);
	double *S3,  *s3;  vectorReader(&S3, "input/fieldSpl", shot.name, concat(field_name ,".spl14"), suc);	hipMalloc((void **) &s3,  dimRZ); 
	double *S4,  *s4;  vectorReader(&S4, "input/fieldSpl", shot.name, concat(field_name ,".spl21"), suc);	hipMalloc((void **) &s4,  dimRZ); 
	double *S5,  *s5;  vectorReader(&S5, "input/fieldSpl", shot.name, concat(field_name ,".spl22"), suc);	hipMalloc((void **) &s5,  dimRZ); 
	double *S6,  *s6;  vectorReader(&S6, "input/fieldSpl", shot.name, concat(field_name ,".spl23"), suc);	hipMalloc((void **) &s6,  dimRZ); 
	double *S7,  *s7;  vectorReader(&S7, "input/fieldSpl", shot.name, concat(field_name ,".spl24"), suc);	hipMalloc((void **) &s7,  dimRZ);
	double *S8,  *s8;  vectorReader(&S8, "input/fieldSpl", shot.name, concat(field_name ,".spl31"), suc);	hipMalloc((void **) &s8,  dimRZ); 
	double *S9,  *s9;  vectorReader(&S9, "input/fieldSpl", shot.name, concat(field_name ,".spl32"), suc);	hipMalloc((void **) &s9,  dimRZ); 
	double *S10, *s10; vectorReader(&S10,"input/fieldSpl", shot.name, concat(field_name ,".spl33"), suc);	hipMalloc((void **) &s10,  dimRZ);
	double *S11, *s11; vectorReader(&S11,"input/fieldSpl", shot.name, concat(field_name ,".spl34"), suc);	hipMalloc((void **) &s11,  dimRZ); 
	double *S12, *s12; vectorReader(&S12,"input/fieldSpl", shot.name, concat(field_name ,".spl41"), suc);	hipMalloc((void **) &s12,  dimRZ);
	double *S13, *s13; vectorReader(&S13,"input/fieldSpl", shot.name, concat(field_name ,".spl42"), suc);	hipMalloc((void **) &s13,  dimRZ); 
	double *S14, *s14; vectorReader(&S14,"input/fieldSpl", shot.name, concat(field_name ,".spl43"), suc);	hipMalloc((void **) &s14,  dimRZ); 
	double *S15, *s15; vectorReader(&S15,"input/fieldSpl", shot.name, concat(field_name ,".spl44"), suc);	hipMalloc((void **) &s15,  dimRZ);
    
	size_t dimB = 16*sizeof(double*);		
	double *S_PTR[16];	double **s_ptr;	hipMalloc((void **) &s_ptr,  dimB);     

	S_PTR[0]  = s0; 	S_PTR[1]  = s1 ;	S_PTR[2]  = s2; 	S_PTR[3]  = s3;
	S_PTR[4]  = s4; 	S_PTR[5]  = s5; 	S_PTR[6]  = s6; 	S_PTR[7]  = s7;
	S_PTR[8]  = s8; 	S_PTR[9]  = s9; 	S_PTR[10] = s10;	S_PTR[11] = s11;
	S_PTR[12] = s12;	S_PTR[13] = s13;	S_PTR[14] = s14;	S_PTR[15] = s15;

	if (suc[0] == 1){
		hipMemcpy(s0, S0, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s1, S1, dimRZ, hipMemcpyHostToDevice);	
		hipMemcpy(s2, S2, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s3, S3, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s4, S4, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s5, S5, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s6, S6, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s7, S7, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s8, S8, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s9, S9, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s10, S10, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s11, S11, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s12, S12, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s13, S13, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s14, S14, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s15, S15, dimRZ, hipMemcpyHostToDevice);			
		hipMemcpy(s_ptr, S_PTR, dimB, hipMemcpyHostToDevice);   
		free(S0);	free(S1);	free(S2);	free(S3);
		free(S4);	free(S5);	free(S6);	free(S7);	
		free(S8);	free(S9);	free(S10);	free(S11);	
		free(S12);	free(S13);	free(S14);	free(S15);
	}
	
	if (shot.debug == 1)
		for (int i=0;i<10;i++){
			printf("spline s0 %d %lf\n",i,S0[i]);
		}
	}
	
	*return_s_ptr = s_ptr; 
	return suc[0];	
    
}

int magnetic_field_read_and_init(shot_prop shot, double ***return_br_ptr, double ***return_bz_ptr, double ***return_bt_ptr, int dimRZ){    
    
	size_t dimB = 16*sizeof(double*);	
	double *BR_PTR[16];	double **br_ptr;	hipMalloc((void **) &br_ptr,  dimB); 
	double *BT_PTR[16];	double **bt_ptr;	hipMalloc((void **) &bt_ptr,  dimB); 
	double *BZ_PTR[16];	double **bz_ptr;	hipMalloc((void **) &bz_ptr,  dimB);     

	int s;    
	s = spline_read_and_init(shot, "brad", &br_ptr, dimRZ);    
	s = spline_read_and_init(shot, "bz",   &bz_ptr, dimRZ);    
	s = spline_read_and_init(shot, "btor", &bt_ptr, dimRZ);     
    
	*return_br_ptr = br_ptr;
	*return_bz_ptr = bz_ptr;
	*return_bt_ptr = bt_ptr;

	return s;
}
int electric_field_read_and_init(shot_prop shot, double ***return_er_ptr, double ***return_ez_ptr, double ***return_et_ptr, int dimRZ){    
    
	size_t dimB = 16*sizeof(double*);	
	double *ER_PTR[16];	double **er_ptr;	hipMalloc((void **) &er_ptr,  dimB); 
	double *ET_PTR[16];	double **et_ptr;	hipMalloc((void **) &et_ptr,  dimB); 
	double *EZ_PTR[16];	double **ez_ptr;	hipMalloc((void **) &ez_ptr,  dimB);     

	int s;   

	s = spline_read_and_init(shot, "erad", &er_ptr, dimRZ);    
	s = spline_read_and_init(shot, "ez",   &ez_ptr, dimRZ);    
	s = spline_read_and_init(shot, "etor", &et_ptr, dimRZ);     
    
	*return_er_ptr = er_ptr;
	*return_ez_ptr = ez_ptr;
	*return_et_ptr = et_ptr;

	return s;
}

int main(int argc, char *argv[]){
    //! @param shotname name of shot folder input folder (8714,11344,11347)	
    
    shot_prop shot;
    beam_prop beam;
	
	if (argc > 1)	shot.name = argv[1];	
	if (argc > 2)	shot.runnumber = atoi(argv[2]);
	if (argc > 3)	beam.matter = argv[3];			
	if (argc > 4)	beam.energy = atof(argv[4]);    
	if (argc > 5)	beam.vertical_deflation = atof(argv[5]);    
	if (argc > 6)	beam.diameter = atof(argv[6]);
	if (argc > 7)	beam.detector_R = atof(argv[7]);
    
	beam.mass = get_mass(beam.matter);
	printf("shotname: %s\n",shot.name);  
		
	int NX;
	int max_blocks;
	if (argc > 8)	max_blocks = atoi(argv[8])/shot.block_number+1;    
		else	max_blocks=shot.block_size;	
        
    if (argc > 9) shot.electric_field_module = atof(argv[9]);     
    
    if (argc > 10){ 
        shot.step_host = atof(argv[10]); 
        shot.step_device = 1;
    }
    
    if (argc > 11) shot.step_device = atof(argv[11]); 
	
    if (argc > 12) shot.debug = atof(argv[12]); 
    
	NX = shot.block_number * max_blocks;
	
	if ($3DINPUTPROF == 1){
        double *XR;
		NX = vectorReader0(&XR, "input/manual_profile/rad.dat");
        max_blocks = NX / shot.block_number+1;
        shot.block_size = NX;
	}	
		
	char* folder_out=concat("results/", shot.name);
	
	set_cuda();

	// set timestamp
	time_t rawtime;
	struct tm *info;
	char timestamp[80];
	sprintf(timestamp, "%d", shot.runnumber);

	// coords	
	double *X_PTR[3], **x_ptr;
	double *V_PTR[3], **v_ptr;
	size_t dimXP = 3*sizeof(double*);

	double *XR,  *xr; 
	double *XZ,  *xz;
	double *XT,  *xt;

	double *VR,  *vr; 
	double *VZ,  *vz;
	double *VT,  *vt;	
	
	printf("=============================\n");
	printf("Number of blocks (threads): %d\n", max_blocks);
	printf("Block size: %d\n", shot.block_size);
	printf("Number of particles: %d\n", NX);
	printf("Max steps on device (GPU): %d\n", shot.step_device);
	printf("Max steps on host (HDD): %d\n", shot.step_host);
	
	XR = (double*)malloc(sizeof(double)*NX);
	XZ = (double*)malloc(sizeof(double)*NX);
	XT = (double*)malloc(sizeof(double)*NX);

	VR = (double*)malloc(sizeof(double)*NX);
	VZ = (double*)malloc(sizeof(double)*NX);
	VT = (double*)malloc(sizeof(double)*NX);

	// phys. constants
	double eperm;
	eperm = 1.60217656535e-19/1.66053892173e-27/beam.mass;

	beamIn(XR, XZ, XT, VR, VZ, VT, beam.energy, eperm, NX, shot.name, beam.diameter, beam.toroidal_deflation, beam.vertical_deflation);

	//! position and velocity array allocation
	size_t dimX = NX * sizeof(double);

	hipMalloc((void **) &xr,  dimX); 
	hipMalloc((void **) &xz,  dimX); 
	hipMalloc((void **) &xt,  dimX); 
	hipMalloc((void **) &x_ptr,  dimXP); 

	hipMalloc((void **) &vr,  dimX); 
	hipMalloc((void **) &vz,  dimX); 
	hipMalloc((void **) &vt,  dimX); 
	hipMalloc((void **) &v_ptr,  dimXP); 

	//! coords pointers
	X_PTR[0] = xr;
	X_PTR[1] = xz;
	X_PTR[2] = xt;

	V_PTR[0] = vr;
	V_PTR[1] = vz;
	V_PTR[2] = vt;
	
	//! grid pointers
	double *G_PTR[2];
	double **g_ptr;
	size_t dimG = 2*sizeof(double*);	
	hipMalloc((void **) &g_ptr,  dimG); 
	double *RG, *rg;
	double *ZG, *zg;

	// size definitions

	//! R-grid points

	int NR = vectorReader(&RG, "input/fieldSpl", shot.name, "r.spline");
	/*if ($ELM == 1){
		NR = vectorReader(&RG, "field/cuda/ipol/r.spline");
		printf("ELM mode\n");
	}else{
		NR = vectorReader(&RG, "input/fieldSpl", shot.name, "r.spline");
		printf("ELM-free mode\n");
	}*/
	size_t dimR = NR * sizeof(double);
	hipMalloc((void **) &rg,  dimR); 
	
	//! Z-grid points
	int NZ = vectorReader(&ZG, "input/fieldSpl", shot.name, "z.spline");
	/*if ($ELM == 1){
		NZ = vectorReader(&ZG, "field/cuda/ipol/z.spline");
	}else{
		NZ = vectorReader(&ZG, "input/fieldSpl", shot.name, "z.spline");
	}*/

	size_t dimZ = NZ * sizeof(double);
	size_t dimRZ = (NR-1) * (NZ-1) * sizeof(double);
	hipMalloc((void **) &zg,  dimZ); 

   	// grid pointer
	G_PTR[0] = rg;
	G_PTR[1] = zg;

	//! MAGN. FIELD (HOST, device) ALLOCATION          
    
    double **br_ptr, **bz_ptr, **bt_ptr;
    double **er_ptr, **ez_ptr, **et_ptr;
    
	int magnetic_field_loaded = magnetic_field_read_and_init(shot, &br_ptr,&bz_ptr,&bt_ptr, dimRZ);
	
	if (shot.electric_field_module){
		shot.electric_field_module = electric_field_read_and_init(shot, &er_ptr,&ez_ptr,&et_ptr, dimRZ);
    }
	
	// temporary test data
	double *TMP, *tmp;
	TMP = (double *)malloc(dimR);	hipMalloc((void **) &tmp,  dimR); 

	// field direction on first
	double *BD1, *bd1;	hipMalloc((void **) &bd1,  dimX); 	BD1 = (double *)malloc(dimX);
	double *BD2, *bd2;	hipMalloc((void **) &bd2,  dimX);	BD2 = (double *)malloc(dimX);

	//! CUDA profiler START
	hipProfilerStart();
	
	//! MEMCOPY (HOST2device)

	//! GRID COORDS	
	hipMemcpy(rg, RG, dimR, hipMemcpyHostToDevice);
	hipMemcpy(zg, ZG, dimZ, hipMemcpyHostToDevice);
	hipMemcpy(g_ptr, G_PTR, dimG, hipMemcpyHostToDevice);
	


	//! ION COORDS (HOST2device)
	hipMemcpy(x_ptr, X_PTR, dimXP, hipMemcpyHostToDevice);	

	//! ION SPEEDS (HOST2device)
	hipMemcpy(v_ptr, V_PTR, dimXP, hipMemcpyHostToDevice);
	
	// EXECUTION
	addData1(XR,NX,folder_out,timestamp,"t_rad.dat");
	addData1(XZ,NX,folder_out,timestamp,"t_z.dat");
	addData1(XZ,NX,folder_out,timestamp,"t_tor.dat");
	addData1(VR,NX,folder_out,timestamp,"t_vrad.dat");
	addData1(VZ,NX,folder_out,timestamp,"t_vz.dat");
	addData1(VZ,NX,folder_out,timestamp,"t_vtor.dat");

	//! Set CUDA timer 
	hipEvent_t start, stop;
	float runtime;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    if (shot.debug == 1){
        printf("ionV:  0.\t %lf\t %lf\t %lf\n",VR[0],VZ[0],VT[0]);
        printf("ionX:  0.\t %lf\t %lf\t %lf\n",XR[0],XZ[0],XT[0]);
        printf("ionX:  1.\t %lf\t %lf\t %lf\n",XR[1],XZ[1],XT[1]);
        
        printf("----------------------------------------------------------\n");
        printf("ion:  0.\t %lf\t %lf\t %lf\n",XR[0],XZ[0],XT[0]);
        printf("----------------------------------------------------------\n");
        for(int i=1; i<20; i++){
            printf("ion: %2d.\t %le\t %le\t %le\n",i,XR[i],XZ[i],XT[i]);
        }
        printf("----------------------------------------------------------\n");
    }
	// BANANA
	if (BANANA==1){
		printf("BANANA CTRL\n");
		// ION COORDS (HOST2device)
		hipMemcpy(xr, XR, dimX, hipMemcpyHostToDevice);
		hipMemcpy(xz, XZ, dimX, hipMemcpyHostToDevice);
		hipMemcpy(xt, XT, dimX, hipMemcpyHostToDevice);
		//hipMemcpy(x_ptr, X_PTR, dimXP, hipMemcpyHostToDevice);	

		// ION SPEEDS (HOST2device)
		hipMemcpy(vr, VR, dimX, hipMemcpyHostToDevice);
		hipMemcpy(vz, VZ, dimX, hipMemcpyHostToDevice);
		hipMemcpy(vt, VT, dimX, hipMemcpyHostToDevice);
				
		banCtrl <<< max_blocks, shot.block_size >>> (NR,NZ,br_ptr,bz_ptr,bt_ptr,g_ptr,x_ptr,bd1,bd2);
		hipMemcpy(BD1, bd1, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(BD2, bd2, dimX, hipMemcpyDeviceToHost);
		addData1(BD1,NX,folder_out,timestamp,"d_b1.dat");
		addData1(BD2,NX,folder_out,timestamp,"d_b2.dat");
		addData1(VR,NX,folder_out,timestamp,"d_vr.dat");
		addData1(VZ,NX,folder_out,timestamp,"d_vz.dat");
		addData1(VT,NX,folder_out,timestamp,"d_vt.dat");
	}	
	
	for (int step_i=0;step_i<shot.step_host;step_i++){
		
		// ION COORDS (HOST2device)
		hipMemcpy(xr, XR, dimX, hipMemcpyHostToDevice);
		hipMemcpy(xz, XZ, dimX, hipMemcpyHostToDevice);
		hipMemcpy(xt, XT, dimX, hipMemcpyHostToDevice);
		//hipMemcpy(x_ptr, X_PTR, dimXP, hipMemcpyHostToDevice);	

		// ION SPEEDS (HOST2device)
		hipMemcpy(vr, VR, dimX, hipMemcpyHostToDevice);
		hipMemcpy(vz, VZ, dimX, hipMemcpyHostToDevice);
		hipMemcpy(vt, VT, dimX, hipMemcpyHostToDevice);
		//hipMemcpy(v_ptr, V_PTR, dimXP, hipMemcpyHostToDevice);
		
		
		// CUDA CODE, timer and Error catch	
		//ERRORCHECK();
		hipEventRecord(start, 0);
		if (shot.electric_field_module){
			printf("electric_field_module ON\n");            
			ctrl <<< max_blocks, shot.block_size >>> (NR,NZ,br_ptr,bz_ptr,bt_ptr,er_ptr,ez_ptr,et_ptr,g_ptr,x_ptr,v_ptr,tmp,eperm,beam.detector_R,shot.step_device);
		}else{
			ctrl <<< max_blocks, shot.block_size >>> (NR,NZ,br_ptr,bz_ptr,bt_ptr,g_ptr,x_ptr,v_ptr,tmp,eperm,beam.detector_R,shot.step_device);
		}
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		ERRORCHECK();

		// ION COORDS (device2HOST)
		hipMemcpy(XR, xr, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(XZ, xz, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(XT, xt, dimX, hipMemcpyDeviceToHost);
		//ERRORCHECK();
		// ION SPEEDS (device2HOST)
		hipMemcpy(VR, vr, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(VZ, vz, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(VT, vt, dimX, hipMemcpyDeviceToHost);
		//ERRORCHECK();
		// Save data to files
		printf("Step\t%d/%d\n",step_i,shot.step_host);
		addData1(XR,NX,folder_out,timestamp,"t_rad.dat");
		addData1(XZ,NX,folder_out,timestamp,"t_z.dat");
		addData1(XZ,NX,folder_out,timestamp,"t_tor.dat");
		addData1(VR,NX,folder_out,timestamp,"t_vrad.dat");
		addData1(VZ,NX,folder_out,timestamp,"t_vz.dat");
		addData1(VZ,NX,folder_out,timestamp,"t_vtor.dat");
		
        if (shot.debug == 1){
            printf("Xion:  0.\t %lf\t %lf\t %lf\n",XR[0],XZ[0],XT[0]);
            printf("Xion:  1.\t %lf\t %lf\t %lf\n",XR[1],XZ[1],XT[1]);
            printf("Vion:  0.\t %lf\t %lf\t %lf\n",VR[0],VZ[0],VT[0]);
        }
        
		if (shot.step_host > 1){            
			for (int i = 1; (i < NX && XR[i] == beam.detector_R); i++){;
				if (i == NX-1) shot.step_host = step_i;
			}
        }
	}	
	
	// Get CUDA timer 
	hipEventElapsedTime(&runtime, start, stop);
	printf ("Time for the kernel: %f s\n", runtime/1000.0);

	//! MEMCOPY (device2HOST)
	hipMemcpy(TMP, tmp, dimR, hipMemcpyDeviceToHost);
	if(TMP[0]!=42.24){
		printf("\n+---	-------------------+\n | Fatal error in running. | \n | The CUDA did not run well. |\n+-----------------------+\n");
	}else{
		printf("\n	Memcopy OK.\n");
	}
	
	if (shot.debug == 1)
		for (int i=0;i<10;i++) {
			printf("TMP%d\t%lf\n",i,TMP[i]);
		}
	}
	//! CUDA profiler STOP
	hipProfilerStop();

	//! Save data to files
	saveData1(XR,NX,folder_out,timestamp,"rad.dat");
	saveData1(XZ,NX,folder_out,timestamp,"z.dat");
	saveData1(XT,NX,folder_out,timestamp,"tor.dat");
	saveData1(VR,NX,folder_out,timestamp,"vrad.dat");
	saveData1(VZ,NX,folder_out,timestamp,"vz.dat");
	saveData1(VT,NX,folder_out,timestamp,"vtor.dat");	
	
	saveDataHT(concat("Shot ID: ",shot.name),folder_out,timestamp);
	saveDataHT(concat("Run ID:  ",timestamp),folder_out,timestamp);
	saveDataHT("-----------------------------------",folder_out,timestamp);
	if(BANANA){
		saveDataHT("BANANA ORBITS",folder_out,timestamp);
	}else{		
		saveDataHT("ABP ION TRAJECTORIES",folder_out,timestamp);
		if(RADIONS){
			saveDataHT("(Real ionization position)",folder_out,timestamp); 
			if($3DINPUTPROF==1){
				saveDataHT("(3D input)",folder_out,timestamp);			
            }else if($RENATE==110){
				saveDataHT("(TS + Renate 1.1.0)",folder_out,timestamp);
			}
			
		}else{
			saveDataHT("(R=const ionization)",folder_out,timestamp);
		}
	}
	saveDataHT("-----------------------------------",folder_out,timestamp);
	saveDataH("Beam energy","keV",beam.energy,folder_out,timestamp);
	saveDataH("Atomic mass","AMU",beam.mass,folder_out,timestamp);
	saveDataH("Beam diameter","mm",beam.diameter,folder_out,timestamp);
	saveDataH2("Deflation (toroidal/vertical)","°",beam.toroidal_deflation,beam.vertical_deflation,folder_out,timestamp);
	if(!RADIONS&&!BANANA){	
		saveDataH("Ion. position (R)","m",R_midions,folder_out,timestamp);
	}
	
	saveDataH("Number of ions","",NX,folder_out,timestamp);
	saveDataHT("-----------------------------------",folder_out,timestamp);
	
	saveDataH("Detector position (R)","m",beam.detector_R,folder_out,timestamp);
	
	saveDataHT("-----------------------------------",folder_out,timestamp);
	
	saveDataH("Timestep","s",dt,folder_out,timestamp);	
	
	saveDataHT("-----------------------------------",folder_out,timestamp);
	
	saveDataH("Kernel runtime", "s", runtime/1000.0,folder_out,timestamp);
	saveDataHT("-----------------------------------",folder_out,timestamp);
	saveDataH("Number of blocks (threads)", "", max_blocks,folder_out,timestamp);
	saveDataH("Block size", "", shot.block_size,folder_out,timestamp);
	saveDataH("Length of a loop", "", shot.step_device,folder_out,timestamp);
	saveDataH("Number of loops", "", shot.step_host,folder_out,timestamp);		

	printf("\nData folder: %s/%s\n\n",folder_out,timestamp);

	//! Free CUDA
	hipFree(x_ptr);	hipFree(xr);	hipFree(xz);	hipFree(xt);
	hipFree(g_ptr);	hipFree(rg);	hipFree(zg);		
	hipFree(br_ptr);	hipFree(bz_ptr);	hipFree(bt_ptr);
    hipFree(er_ptr);	hipFree(ez_ptr);	hipFree(et_ptr);
    /*
	hipFree(br0);	hipFree(br1);	hipFree(br2);	hipFree(br3);	
	hipFree(br4);	hipFree(br5);	hipFree(br6);	hipFree(br7);	
	hipFree(br8);	hipFree(br9);	hipFree(br10);	hipFree(br11);	
	hipFree(br12);	hipFree(br13);	hipFree(br14);	hipFree(br15);
		
	hipFree(bz0);	hipFree(bz1);	hipFree(bz2);	hipFree(bz3);
	hipFree(bz4);	hipFree(bz5);	hipFree(bz6);	hipFree(bz7);
	hipFree(bz8);	hipFree(bz9);	hipFree(bz10);	hipFree(bz11);
	hipFree(bz12);	hipFree(bz13);	hipFree(bz14);	hipFree(bz15);
	
	hipFree(bt0);	hipFree(bt1);	hipFree(bt2);	hipFree(bt3);	
	hipFree(bt4);	hipFree(bt5);	hipFree(bt6);	hipFree(bt7);	
	hipFree(bt8);	hipFree(bt9);	hipFree(bt10);	hipFree(bt11);	
	hipFree(bt12);	hipFree(bt13);	hipFree(bt14);	hipFree(bt15);	*/

	//! Free RAM
	free(RG);	free(ZG);	
	free(XR);	free(XZ);	free(XT);
	//	free(G_PTR);
	//	free(BR_PTR);	free(BZ_PTR);	free(BT_PTR);	
	
	/*free(BR0);	free(BR1);	free(BR2);	free(BR3);
	free(BR4);	free(BR5);	free(BR6);	free(BR7);	
	free(BR8);	free(BR9);	free(BR10);	free(BR11);	
	free(BR12);	free(BR13);	free(BR14);	free(BR15);
		
	free(BZ0);	free(BZ1);	free(BZ2);	free(BZ3);	
	free(BZ4);	free(BZ5);	free(BZ6);	free(BZ7);	
	free(BZ8);	free(BZ9);	free(BZ10);	free(BZ11);	
	free(BZ12);	free(BZ13);	free(BZ14);	free(BZ15);
	
	free(BT0);	free(BT1);	free(BT2);	free(BT3);
	free(BT4);	free(BT5);	free(BT6);	free(BT7);
	free(BT8);	free(BT9);	free(BT10);	free(BT11);
	free(BT12);	free(BT13);	free(BT14);	free(BT15);		*/
	
	//! FREE TMP variables (RAM, cuda)
	free(TMP);	hipFree(tmp);

	printf("Ready.\n\n");
}

char* concat(const char *s1, const char *s2){
    char *result = (char*)malloc(strlen(s1)+strlen(s2)+1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}
