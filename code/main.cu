#include "hip/hip_runtime.h"
// TAIGA default parameters

#define N_BLOCKS	 1		//! @param N_BLOCKS number of blocks (max 1M)
#define BLOCK_SIZE 	 192 		//! @param BLOCK_SIZE size of blocks (max 192 on Geforce GTS450) (max 768 on Geforce GTS650Ti)

#define $R_defl	2.3			//! radial position of deflection plates in meter -> TOROIDAL DEFLECTION
#define $deflH	 0				//! @param $deflH horizontal deflection in rad (up--down)  
#define $deflV	 0				//! @param $deflV vertical deflection in rad (left--right) -> TOROIDAL DEFLECTION

#define $default_energy   60				//! @param energy in keV
#define $default_mass  7.016004558			//! @param atomic mass in amu

#define $default_diameter 25//4/*e-20*/	  //! @param diameter in mm

#define dt	   1e-9			//! @param dt timestep in seconds

#define ERRORCHECK() cErrorCheck(__FILE__, __LINE__)
#define PI 3.141592653589792346
#define ELEMENTARY_CHARGE 1.60217656535e-19
#define AMU 1.66053892173e-27

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#include <string.h>
//#include <filesystem>

#include <hip/hip_runtime_api.h>
#include "cuda/nvToolsExt.h"

#include "main.cuh"
#include "dataio/fieldIn.c"

#if READINPUTPROF == 1
	#include "dataio/beamInFull.c"
#elif RENATE == 110
	#include "dataio/beamInRenate110.c"
#else
	#include "dataio/beamIn.c"
#endif

#include "dataio/beamOut.c"

#include "running/rk4.cu"
#include "running/ipol.cu"
#include "running/cyl2tor.cu"
#include "running/traj.cu"
#include "running/ctrl.cu"

#include "dataio/detectorIn.c"
#include "running/detector_postproc.cu"

int input_init_taiga(int argc, char *argv[], shot_prop *shot, beam_prop *beam){
	int max_blocks;

	if (argc > 1)	shot->name = argv[1];	
	if (argc > 2)	shot->runnumber = atoi(argv[2]);
	if (argc > 3)	beam->matter = argv[3];
	if (argc > 4)	beam->energy = atof(argv[4]);
	if (argc > 5)	beam->vertical_deflation = atof(argv[5]);
	if (argc > 6)	beam->diameter = atof(argv[6]);  

	if (argc > 8)	max_blocks = atoi(argv[8])/shot->block_size+1; 
		else	max_blocks=shot->block_number;

	if (argc > 9) shot->electric_field_module = atof(argv[9]);

	if (argc > 10){ 
		shot->step_host = atof(argv[10]); 
		shot->step_device = 1;
	}
	if (argc > 11)	shot->step_device = atof(argv[11]); 	
	if (argc > 12)	shot->debug = atof(argv[12]); 

	beam->mass = get_mass(beam->matter);
	return max_blocks;   
}

int main(int argc, char *argv[]){
	//! @param shotname name of shot folder input folder (8714,11344,11347)	
	
	shot_prop shot;
	beam_prop beam;
	int max_blocks = input_init_taiga(argc, argv, &shot, &beam);

	size_t dimD = 5 * sizeof(double);
	double *DETECTOR, *detector;
	DETECTOR = (double *)malloc(dimD);	hipMalloc((void **) &detector,  dimD); 
	
	if (argc > 7)	fill_detector(DETECTOR, argv[7]);

	printf("shotname: %s\n",shot.name);  
	printf("detector: [ %lf %lf %lf %lf %lf]\n", DETECTOR[0],DETECTOR[1],DETECTOR[2],DETECTOR[3],DETECTOR[4]);

	int NX = shot.block_size * max_blocks;

	if (READINPUTPROF == 1){
		double *XR;
		NX = vectorReader0(&XR, "input/manual_profile/rad.dat");
		max_blocks = NX / shot.block_size+1;
	}

	char* folder_out=concat("results/", shot.name);
	
	set_cuda();

	// set timestamp
	time_t rawtime;
	struct tm *info;
	char timestamp[80];
	sprintf(timestamp, "%d", shot.runnumber);

	// coords
	double *X_PTR[3], **x_ptr;
	double *V_PTR[3], **v_ptr;
	size_t dimXP = 3*sizeof(double*);

	double *XR,  *xr; 
	double *XZ,  *xz;
	double *XT,  *xt;

	double *VR,  *vr; 
	double *VZ,  *vz;
	double *VT,  *vt;

	printf("=============================\n");
	printf("Number of blocks (threads): %d\n", max_blocks);
	printf("Block size: %d\n", shot.block_size);
	printf("Number of particles: %d\n", NX);
	printf("Max steps on device (GPU): %d\n", shot.step_device);
	printf("Max steps on host (HDD): %d\n", shot.step_host);


	//! position and velocity array allocation
	size_t dimX = shot.block_size * max_blocks * sizeof(double);
	
	XR = (double*)malloc(dimX);
	XZ = (double*)malloc(dimX);
	XT = (double*)malloc(dimX);

	VR = (double*)malloc(dimX);
	VZ = (double*)malloc(dimX);
	VT = (double*)malloc(dimX);

	// phys. constants
	double eperm = ELEMENTARY_CHARGE/ AMU/ beam.mass;

	beamIn(XR, XZ, XT, VR, VZ, VT, beam.energy, eperm, NX, shot.name, beam.diameter, beam.toroidal_deflation, beam.vertical_deflation);

	hipMalloc((void **) &xr,  dimX); 
	hipMalloc((void **) &xz,  dimX); 
	hipMalloc((void **) &xt,  dimX); 
	hipMalloc((void **) &x_ptr,  dimXP); 

	hipMalloc((void **) &vr,  dimX); 
	hipMalloc((void **) &vz,  dimX); 
	hipMalloc((void **) &vt,  dimX); 
	hipMalloc((void **) &v_ptr,  dimXP); 

	//! coords pointers
	X_PTR[0] = xr;
	X_PTR[1] = xz;
	X_PTR[2] = xt;

	V_PTR[0] = vr;
	V_PTR[1] = vz;
	V_PTR[2] = vt;
	
	//! grid pointers
	double *G_PTR[2];
	double **g_ptr;
	size_t dimG = 2*sizeof(double*);	
	hipMalloc((void **) &g_ptr,  dimG); 
	double *RG, *rg;
	double *ZG, *zg;

	// size definitions

	//! R-grid points
	int NR = vectorReader(&RG, "input/fieldSpl", shot.name, "r.spline");
	size_t dimR = NR * sizeof(double);
	hipMalloc((void **) &rg,  dimR); 
	
	//! Z-grid points
	int NZ = vectorReader(&ZG, "input/fieldSpl", shot.name, "z.spline");
	size_t dimZ = NZ * sizeof(double);
	size_t dimRZ = (NR-1) * (NZ-1) * sizeof(double);
	hipMalloc((void **) &zg,  dimZ); 

   	// grid pointer
	G_PTR[0] = rg;
	G_PTR[1] = zg;

	//! MAGN. FIELD (HOST, device) ALLOCATION  
	double **br_ptr, **bz_ptr, **bt_ptr;
	double **er_ptr, **ez_ptr, **et_ptr;
	
	int magnetic_field_loaded = magnetic_field_read_and_init(shot, &br_ptr,&bz_ptr,&bt_ptr, dimRZ);	
	if (shot.electric_field_module)	shot.electric_field_module = electric_field_read_and_init(shot, &er_ptr,&ez_ptr,&et_ptr, dimRZ);
	
	// detector cell id
	size_t dimRint = NX * sizeof(int);
	int *DETCELLID, *detcellid;
	DETCELLID = (int *)malloc(dimRint);	hipMalloc((void **) &detcellid,  dimRint); 	
	
	// temporary test data
	size_t dimService = 10 * sizeof(double);
	double *SERVICE_VAR, *service_var;
	SERVICE_VAR = (double *)malloc(dimService);	hipMalloc((void **) &service_var,  dimService); 

	//! CUDA profiler START
	hipProfilerStart();
	
	//! MEMCOPY (HOST2device)

	//! GRID COORDS	
	hipMemcpy(rg, RG, dimR, hipMemcpyHostToDevice);
	hipMemcpy(zg, ZG, dimZ, hipMemcpyHostToDevice);
	hipMemcpy(g_ptr, G_PTR, dimG, hipMemcpyHostToDevice);

	//! ION COORDS (HOST2device)
	hipMemcpy(x_ptr, X_PTR, dimXP, hipMemcpyHostToDevice);	

	//! ION SPEEDS (HOST2device)
	hipMemcpy(v_ptr, V_PTR, dimXP, hipMemcpyHostToDevice);

	//! DETECTOR COORDS (HOST2device)
	hipMemcpy(detector, DETECTOR, dimD, hipMemcpyHostToDevice);
	
	// OUTPUT INIT
	addData1(XR,NX,folder_out,timestamp,"t_rad.dat");
	addData1(XZ,NX,folder_out,timestamp,"t_z.dat");
	addData1(XT,NX,folder_out,timestamp,"t_tor.dat");
	addData1(VR,NX,folder_out,timestamp,"t_vrad.dat");
	addData1(VZ,NX,folder_out,timestamp,"t_vz.dat");
	addData1(VT,NX,folder_out,timestamp,"t_vtor.dat");

	//! Set CUDA timer 
	hipEvent_t start, stop;
	float runtime;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (shot.debug == 1)	debug_message_init(XR, XZ, XT, VR, VZ, VT);
	
	for (int step_i=0;step_i<shot.step_host;step_i++){
		
		// ION COORDS (HOST2device)
		hipMemcpy(xr, XR, dimX, hipMemcpyHostToDevice);
		hipMemcpy(xz, XZ, dimX, hipMemcpyHostToDevice);
		hipMemcpy(xt, XT, dimX, hipMemcpyHostToDevice);
		//hipMemcpy(x_ptr, X_PTR, dimXP, hipMemcpyHostToDevice);	

		// ION SPEEDS (HOST2device)
		hipMemcpy(vr, VR, dimX, hipMemcpyHostToDevice);
		hipMemcpy(vz, VZ, dimX, hipMemcpyHostToDevice);
		hipMemcpy(vt, VT, dimX, hipMemcpyHostToDevice);
		//hipMemcpy(v_ptr, V_PTR, dimXP, hipMemcpyHostToDevice);
				
		//ERRORCHECK();
		
		hipEventRecord(start, 0);
		if (shot.electric_field_module){
			printf("electric_field_module ON\n");
			ctrl <<< max_blocks, shot.block_size >>> (NR,NZ,eperm,br_ptr,bz_ptr,bt_ptr,er_ptr,ez_ptr,et_ptr,g_ptr,x_ptr,v_ptr,detector,detcellid,shot.step_device,service_var);
		}else{
			ctrl <<< max_blocks, shot.block_size >>> (NR,NZ,eperm,br_ptr,bz_ptr,bt_ptr,g_ptr,x_ptr,v_ptr,detector,detcellid,shot.step_device,service_var);
		}
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		ERRORCHECK();

		// ION COORDS (device2HOST)
		hipMemcpy(XR, xr, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(XZ, xz, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(XT, xt, dimX, hipMemcpyDeviceToHost);
		//ERRORCHECK();
		
		// ION SPEEDS (device2HOST)
		hipMemcpy(VR, vr, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(VZ, vz, dimX, hipMemcpyDeviceToHost);
		hipMemcpy(VT, vt, dimX, hipMemcpyDeviceToHost);
		//ERRORCHECK();
		
		// DETCELLID (device2HOST)
		hipMemcpy(DETCELLID, detcellid, dimRint, hipMemcpyDeviceToHost);
		//ERRORCHECK();
		
		// Save data to files
		printf("Step\t%d/%d\n",step_i,shot.step_host);
		addData1(XR,NX,folder_out,timestamp,"t_rad.dat");
		addData1(XZ,NX,folder_out,timestamp,"t_z.dat");
		addData1(XT,NX,folder_out,timestamp,"t_tor.dat");
		addData1(VR,NX,folder_out,timestamp,"t_vrad.dat");
		addData1(VZ,NX,folder_out,timestamp,"t_vz.dat");
		addData1(VT,NX,folder_out,timestamp,"t_vtor.dat");
		
		if (shot.debug == 1)	debug_message_run(XR, XZ, XT, VR, VZ, VT);

		/*if (shot.step_host > 1){
			for (int i = 1; (i < NX && XR[i] == detector); i++){;
				if (i == NX-1) shot.step_host = step_i;
			}
		}*/
	}

	// Get CUDA timer 
	hipEventElapsedTime(&runtime, start, stop);
	printf ("Time for the kernel: %f s\n", runtime/1000.0);

	//! MEMCOPY (device2HOST)
	hipMemcpy(SERVICE_VAR, service_var, dimService, hipMemcpyDeviceToHost);
	if(SERVICE_VAR[0]!=42.24){
		printf("\n +--------------------------+\n | Fatal error in running.    | \n | The CUDA did not run well. |\n +---------------------------+\n");
	}else{
		printf("\n	Memcopy OK.\n");
	}

	/*if (shot.debug == 1){
		for (int i=0;i<10;i++) {
			printf("SERVICE_VAR%d\t%lf\n",i,SERVICE_VAR[i]);
		}
	}*/

	detector_module(x_ptr, detector, detcellid, "test"); //detector_name = "test"

	//! CUDA profiler STOP
	hipProfilerStop();

	//! Save data to files
	saveData1(XR,NX,folder_out,timestamp,"rad.dat");
	saveData1(XZ,NX,folder_out,timestamp,"z.dat");
	saveData1(XT,NX,folder_out,timestamp,"tor.dat");
	saveData1(VR,NX,folder_out,timestamp,"vrad.dat");
	saveData1(VZ,NX,folder_out,timestamp,"vz.dat");
	saveData1(VT,NX,folder_out,timestamp,"vtor.dat");
	saveData1(DETCELLID,NX,folder_out,timestamp,"detcellid.dat");
	
	saveDataHT(concat("Shot ID: ",shot.name),folder_out,timestamp);
	saveDataHT(concat("Run ID:  ",timestamp),folder_out,timestamp);
	saveDataHT("-----------------------------------",folder_out,timestamp);
	saveDataHT(concat("version: r ",SVN_REV),folder_out,timestamp);
	saveDataHT("-----------------------------------",folder_out,timestamp);
		
	saveDataHT("ABP ION TRAJECTORIES",folder_out,timestamp);

	saveDataHT("(Real ionization position)",folder_out,timestamp); 
	if(READINPUTPROF==1){
		saveDataHT("(3D input)",folder_out,timestamp);			
	}else if(RENATE==110){
		saveDataHT("(TS + Renate 1.1.0)",folder_out,timestamp);
	}

	saveDataHT("-----------------------------------",folder_out,timestamp);

	if(!READINPUTPROF){
		saveDataH("Beam energy","keV",beam.energy,folder_out,timestamp);
		saveDataH("Atomic mass","AMU",beam.mass,folder_out,timestamp);
		saveDataH("Beam diameter","mm",beam.diameter,folder_out,timestamp);
		saveDataH2("Deflation (toroidal/vertical)","°",beam.toroidal_deflation,beam.vertical_deflation,folder_out,timestamp);
	}
	
	
	saveDataH("Number of ions","",NX,folder_out,timestamp);
	saveDataHT("-----------------------------------",folder_out,timestamp);
	
	 // DETECTOR
	saveDataH("Detector position (R)","m",DETECTOR[0],folder_out,timestamp);
	saveDataH("Detector position (Z)","m",DETECTOR[1],folder_out,timestamp);
	saveDataH("Detector position (T)","m",DETECTOR[2],folder_out,timestamp);
	saveDataH("Detector angle (Z/R)","°",atan(DETECTOR[3])/PI*180.0,folder_out,timestamp);
	saveDataH("Detector angle (T/R)","°",atan(DETECTOR[4])/PI*180.0,folder_out,timestamp);
	
	saveDataHT("-----------------------------------",folder_out,timestamp);
	
	saveDataH("Timestep","s",dt,folder_out,timestamp);	
	
	saveDataHT("-----------------------------------",folder_out,timestamp);
	
	saveDataH("Kernel runtime", "s", runtime/1000.0,folder_out,timestamp);
	saveDataHT("-----------------------------------",folder_out,timestamp);
	saveDataH("Number of blocks (threads)", "", max_blocks,folder_out,timestamp);
	saveDataH("Block size", "", shot.block_size,folder_out,timestamp);
	saveDataH("Length of a loop", "", shot.step_device,folder_out,timestamp);
	saveDataH("Number of loops", "", shot.step_host,folder_out,timestamp);		

	printf("\nData folder: %s/%s\n\n",folder_out,timestamp);

	//! Free CUDA
	hipFree(x_ptr);	hipFree(xr);	hipFree(xz);	hipFree(xt);
	hipFree(g_ptr);	hipFree(rg);	hipFree(zg);		
	hipFree(br_ptr);	hipFree(bz_ptr);	hipFree(bt_ptr);
	hipFree(er_ptr);	hipFree(ez_ptr);	hipFree(et_ptr);

	//! Free RAM
	free(RG);	free(ZG);	
	free(XR);	free(XZ);	free(XT);
	
	//! FREE SERVICE_VAR variables (RAM, cuda)
	free(SERVICE_VAR);	hipFree(service_var);

	printf("Ready.\n\n");
}


inline void cErrorCheck(const char *file, int line) {
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
	printf("Error: %s\n", hipGetErrorString(err));
	printf(" @ %s: %d\n", file, line);
	exit(-1);
  }
}

int set_cuda(){
	int num_devices, device, max_device;
	hipGetDeviceCount(&num_devices);
	printf("Number of devices: %d\n",num_devices);
	
	if (num_devices > 1) {
		int max_multiprocessors = 0, max_device = 0;
		for (device = 0; device < num_devices; device++) {
			hipDeviceProp_t properties;
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				max_device = device;
			}
		  /*  printf("%d:%s\n",device,&properties.name);
			printf("\tL2Cache:\t%d",	properties.l2CacheSize);
			printf("\tNumber of cores:\t%d",	properties.warpSize);
	
			printf("\tKernels:\t%d",	properties.concurrentKernels);
			printf("\tThreads:\t%d",	properties.maxThreadsPerMultiProcessor);
			printf("\tClock:\t%d",	properties.clockRate/1024);
			printf("\n");*/
		}
		hipSetDevice(max_device);
		for (device = 0; device < num_devices; device++) {
			if(device==max_device) printf("-->");
			hipDeviceProp_t properties;
			hipGetDeviceProperties(&properties, device);
			printf("\t%d:\t%s\n",device,&properties.name);
		}

	}

	hipDeviceProp_t prop;
	hipGetDevice(&max_device);
	hipGetDeviceProperties(&prop, 0) ;  
}

double get_mass(char *s){
	double mass;
	
	if (strcmp(s,"D")==0){
		mass = 2.013553212724;
	}else if (strcmp(s,"Li")==0){
		mass = 7.016004558;
	}else if (strcmp(s,"Na")==0){
		mass = 22.98976928;
	}else if (strcmp(s,"K")==0){
		mass = 39.9639984821;
	}else if (strcmp(s,"H2")==0){
		mass = 2.013553212724;
	}else if (strcmp(s,"Li7")==0){
		mass = 7.016004558;
	}else if (strcmp(s,"Na23")==0){
		mass = 22.98976928;
	}else if (strcmp(s,"K40")==0){
		mass = 39.9639984821;
	}else{
		try{
			mass = atof(s);
		}catch (...){
			mass = (double)$default_mass;
		}
	}
	
	return mass;
}

int spline_read_and_init(shot_prop shot, char* field_name, double ***return_s_ptr, int dimRZ){

	char* spline_folder = "input/fieldSpl";
	int suc[1] = {1};
	
	double *S0,  *s0;  vectorReader(&S0, "input/fieldSpl", shot.name, concat(field_name ,".spl11"), suc);	hipMalloc((void **) &s0,  dimRZ); 
	double *S1,  *s1;  vectorReader(&S1, "input/fieldSpl", shot.name, concat(field_name ,".spl12"), suc);	hipMalloc((void **) &s1,  dimRZ);
	double *S2,  *s2;  vectorReader(&S2, "input/fieldSpl", shot.name, concat(field_name ,".spl13"), suc);	hipMalloc((void **) &s2,  dimRZ);
	double *S3,  *s3;  vectorReader(&S3, "input/fieldSpl", shot.name, concat(field_name ,".spl14"), suc);	hipMalloc((void **) &s3,  dimRZ); 
	double *S4,  *s4;  vectorReader(&S4, "input/fieldSpl", shot.name, concat(field_name ,".spl21"), suc);	hipMalloc((void **) &s4,  dimRZ); 
	double *S5,  *s5;  vectorReader(&S5, "input/fieldSpl", shot.name, concat(field_name ,".spl22"), suc);	hipMalloc((void **) &s5,  dimRZ); 
	double *S6,  *s6;  vectorReader(&S6, "input/fieldSpl", shot.name, concat(field_name ,".spl23"), suc);	hipMalloc((void **) &s6,  dimRZ); 
	double *S7,  *s7;  vectorReader(&S7, "input/fieldSpl", shot.name, concat(field_name ,".spl24"), suc);	hipMalloc((void **) &s7,  dimRZ);
	double *S8,  *s8;  vectorReader(&S8, "input/fieldSpl", shot.name, concat(field_name ,".spl31"), suc);	hipMalloc((void **) &s8,  dimRZ); 
	double *S9,  *s9;  vectorReader(&S9, "input/fieldSpl", shot.name, concat(field_name ,".spl32"), suc);	hipMalloc((void **) &s9,  dimRZ); 
	double *S10, *s10; vectorReader(&S10,"input/fieldSpl", shot.name, concat(field_name ,".spl33"), suc);	hipMalloc((void **) &s10,  dimRZ);
	double *S11, *s11; vectorReader(&S11,"input/fieldSpl", shot.name, concat(field_name ,".spl34"), suc);	hipMalloc((void **) &s11,  dimRZ); 
	double *S12, *s12; vectorReader(&S12,"input/fieldSpl", shot.name, concat(field_name ,".spl41"), suc);	hipMalloc((void **) &s12,  dimRZ);
	double *S13, *s13; vectorReader(&S13,"input/fieldSpl", shot.name, concat(field_name ,".spl42"), suc);	hipMalloc((void **) &s13,  dimRZ); 
	double *S14, *s14; vectorReader(&S14,"input/fieldSpl", shot.name, concat(field_name ,".spl43"), suc);	hipMalloc((void **) &s14,  dimRZ); 
	double *S15, *s15; vectorReader(&S15,"input/fieldSpl", shot.name, concat(field_name ,".spl44"), suc);	hipMalloc((void **) &s15,  dimRZ);
	
	size_t dimB = 16*sizeof(double*);		
	double *S_PTR[16];	double **s_ptr;	hipMalloc((void **) &s_ptr,  dimB);	 

	S_PTR[0]  = s0; 	S_PTR[1]  = s1 ;	S_PTR[2]  = s2; 	S_PTR[3]  = s3;
	S_PTR[4]  = s4; 	S_PTR[5]  = s5; 	S_PTR[6]  = s6; 	S_PTR[7]  = s7;
	S_PTR[8]  = s8; 	S_PTR[9]  = s9; 	S_PTR[10] = s10;	S_PTR[11] = s11;
	S_PTR[12] = s12;	S_PTR[13] = s13;	S_PTR[14] = s14;	S_PTR[15] = s15;

	if (suc[0] == 1){
		hipMemcpy(s0, S0, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s1, S1, dimRZ, hipMemcpyHostToDevice);	
		hipMemcpy(s2, S2, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s3, S3, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s4, S4, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s5, S5, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s6, S6, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s7, S7, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s8, S8, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s9, S9, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s10, S10, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s11, S11, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s12, S12, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s13, S13, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s14, S14, dimRZ, hipMemcpyHostToDevice);
		hipMemcpy(s15, S15, dimRZ, hipMemcpyHostToDevice);			
		hipMemcpy(s_ptr, S_PTR, dimB, hipMemcpyHostToDevice);   
		free(S0);	free(S1);	free(S2);	free(S3);
		free(S4);	free(S5);	free(S6);	free(S7);	
		free(S8);	free(S9);	free(S10);	free(S11);	
		free(S12);	free(S13);	free(S14);	free(S15);
	}
	
	if (shot.debug == 1){
		for (int i=0;i<10;i++){
			printf("spline s0 %d %lf\n",i,S0[i]);
		}
	}
	
	*return_s_ptr = s_ptr; 
	return suc[0];	

}

int magnetic_field_read_and_init(shot_prop shot, double ***return_br_ptr, double ***return_bz_ptr, double ***return_bt_ptr, int dimRZ){

	size_t dimB = 16*sizeof(double*);	
	double *BR_PTR[16];	double **br_ptr;	hipMalloc((void **) &br_ptr,  dimB); 
	double *BT_PTR[16];	double **bt_ptr;	hipMalloc((void **) &bt_ptr,  dimB); 
	double *BZ_PTR[16];	double **bz_ptr;	hipMalloc((void **) &bz_ptr,  dimB);

	int s;
	s = spline_read_and_init(shot, "brad", &br_ptr, dimRZ);
	s = spline_read_and_init(shot, "bz",   &bz_ptr, dimRZ);
	s = spline_read_and_init(shot, "btor", &bt_ptr, dimRZ);

	*return_br_ptr = br_ptr;
	*return_bz_ptr = bz_ptr;
	*return_bt_ptr = bt_ptr;

	return s;
}

void fill_detector(double *DETECTOR, char* values){

	char *el; 
	el = strtok(values,",");	DETECTOR[0] = strtod (el, NULL);
	el = strtok(NULL,",");	DETECTOR[1] = strtod (el, NULL);
	el = strtok(NULL,",");	DETECTOR[2] = tan(strtod (el, NULL) * PI/180.0);
	el = strtok(NULL,",");	DETECTOR[3] = tan(strtod (el, NULL) * PI/180.0);

}


void process_detector(int *detcellid, double **x_ptr){
	
}

int electric_field_read_and_init(shot_prop shot, double ***return_er_ptr, double ***return_ez_ptr, double ***return_et_ptr, int dimRZ){	

	size_t dimB = 16*sizeof(double*);	
	double *ER_PTR[16];	double **er_ptr;	hipMalloc((void **) &er_ptr,  dimB);
	double *ET_PTR[16];	double **et_ptr;	hipMalloc((void **) &et_ptr,  dimB);
	double *EZ_PTR[16];	double **ez_ptr;	hipMalloc((void **) &ez_ptr,  dimB);

	int s;   

	s = spline_read_and_init(shot, "erad", &er_ptr, dimRZ);
	s = spline_read_and_init(shot, "ez",   &ez_ptr, dimRZ);
	s = spline_read_and_init(shot, "etor", &et_ptr, dimRZ);

	*return_er_ptr = er_ptr;
	*return_ez_ptr = ez_ptr;
	*return_et_ptr = et_ptr;

	return s;
}

//

char* concat(const char *s1, const char *s2){
	char *result = (char*)malloc(strlen(s1)+strlen(s2)+1);
	strcpy(result, s1);
	strcat(result, s2);
	return result;
}

// DEBUG

void debug_message_init(double* XR, double* XZ, double* XT, double* VR, double* VZ, double* VT){
		printf("ionV:  0.\t %lf\t %lf\t %lf\n",VR[0],VZ[0],VT[0]);
		printf("ionX:  0.\t %lf\t %lf\t %lf\n",XR[0],XZ[0],XT[0]);
		printf("ionX:  1.\t %lf\t %lf\t %lf\n",XR[1],XZ[1],XT[1]);
		
		printf("----------------------------------------------------------\n");
		printf("ion:  0.\t %lf\t %lf\t %lf\n",XR[0],XZ[0],XT[0]);
		printf("----------------------------------------------------------\n");
		for(int i=1; i<20; i++){
			printf("ion: %2d.\t %le\t %le\t %le\n",i,XR[i],XZ[i],XT[i]);
		}
		printf("----------------------------------------------------------\n");
  
}

void debug_message_run(double* XR, double* XZ, double* XT, double* VR, double* VZ, double* VT){
			printf("Xion:  0.\t %lf\t %lf\t %lf\n",XR[0],XZ[0],XT[0]);
			printf("Xion:  1.\t %lf\t %lf\t %lf\n",XR[1],XZ[1],XT[1]);
			printf("Vion:  0.\t %lf\t %lf\t %lf\n",VR[0],VZ[0],VT[0]);
}
