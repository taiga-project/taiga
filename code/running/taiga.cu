#include "hip/hip_runtime.h"
__global__ void taiga(taiga_globals g, taiga_commons s, double *service_var){
//(double timestep, int NR, int NZ, double eperm, double **spline_brad, double **spline_bz, double **spline_btor, double **spline_grid, double **position_all, double **speed_all, double *detector_geometry, int *detcellid, int N_step, double *service_var, int step_i){
    // thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    service_var[6] = 3.1415926535897932456;
    
    //if (s->step_counter == 0)    g->detcellid[idx] = -1;
    
    if (g.detcellid[idx] == -1){
        taiga_locals l;
        //hipMalloc((void **) &(l.coords),  6*sizeof(double));
        l.coords[0] = g.rad[idx];
        l.coords[1] = g.z[idx];
        l.coords[2] = g.tor[idx];
        l.coords[3] = g.vrad[idx];
        l.coords[4] = g.vz[idx];
        l.coords[5] = g.vtor[idx];
        
        l.detcellid = g.detcellid[idx];
        g.detcellid[idx] = traj(l, s);
        
        service_var[6] = s.brad[0][0];
        service_var[7] = (double)s.grid_size[0]+(double)s.grid_size[1]/1000;
        service_var[3] = s.spline_rgrid[0];
        service_var[4] = s.spline_zgrid[0];
        service_var[5] = s.spline_zgrid[1];
        
        g.rad[idx]  = l.coords[0];
        g.z[idx]    = l.coords[1];
        g.tor[idx]  = l.coords[2];
        g.vrad[idx] = l.coords[3];
        g.vz[idx]   = l.coords[4];
        g.vtor[idx] = l.coords[5];
    }
    service_var[0] = 42.24;
}

__global__ void cuda_service_test(/*taiga_globals g, taiga_commons s,*/ double *service_var){
    service_var[9] = 3.1415926535897932456;
}
