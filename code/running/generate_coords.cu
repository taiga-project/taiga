#include "hip/hip_runtime.h"
__device__ double device_linear_interpolate(double *x_vector, int x_length, double *y_vector, int y_length, double x_value){
    int i;       
    for (i=1; (i<x_length) && (x_vector[i-1]>x_value); i++);    
    if(i>1){--i;}else{i=1;}    
    return y_vector[i] - (y_vector[i]-y_vector[i-1])*(x_value-x_vector[i-1])/(x_vector[i]-x_vector[i-1]);
}

__global__ void generate_coords(double beam_diameter, double beam_energy, double beam_vertical_deflection, double beam_toroidal_deflection,
                                double **position_all, double **speed_all, double eperm, int *prof_size, double *prof_r, double *prof_d, double *profx_r, double *profx_d){
    // thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int i;
    double Vabs, ionisation_yeald, xsec_rad, xsec_ang;
    double XR, XZ, XT;
    
    Vabs = sqrt(2*beam_energy*1000*eperm);
    
    /* cross section normalisation */
    if (prof_size[1] > 0){
        for (i=0; i<prof_size[1]; i++){
            profx_d[i] /= profx_r[i];
        }
    }
    
    /* initialize random generator */
    hiprandState state;
	hiprand_init((unsigned long long)clock() + idx, 0, 0, &state);	
	    
    /* set position of particles */
    do{
        ionisation_yeald = hiprand_uniform_double(&state);
        XR = device_linear_interpolate(prof_d, prof_size[0], prof_r, prof_size[0], ionisation_yeald);
        position_all[0][idx] = XR;
    }while (isnan(XR)||XR<0);
    do{
        //if (prof_size[1] <= 0){
            XZ = (hiprand_uniform_double(&state)-0.5)*beam_diameter;
            XT = (hiprand_uniform_double(&state)-0.5)*beam_diameter;
            position_all[1][idx] = XZ;
            position_all[2][idx] = XT;
        /*}else{
            ionisation_yeald = hiprand_uniform_double(&state);
            xsec_ang = hiprand_uniform_double(&state)*2*PI;
            xsec_rad = linear_interpolate(profx_d, prof_size[1], profx_r, prof_size[1], ionisation_yeald)*(beam_diameter/2);
            XZ[i]= sin(xsec_ang) * xsec_rad;
            XT[i]= cos(xsec_ang) * xsec_rad;
        }*/
    }while ((XZ*XZ+XT*XT)>=(beam_diameter/2)*(beam_diameter/2));
    
    
    // deflection 
    position_all[1][idx] += tan(beam_vertical_deflection) * ($R_defl - XR);
    position_all[2][idx] += tan(beam_toroidal_deflection) * ($R_defl - XR);
    
    // set velocity of particles
    speed_all[0][idx] = -Vabs*cos(beam_vertical_deflection)*cos(beam_toroidal_deflection);
    speed_all[1][idx] =  Vabs*sin(beam_vertical_deflection);
    speed_all[2][idx] =  Vabs*cos(beam_vertical_deflection)*sin(beam_toroidal_deflection);
}
