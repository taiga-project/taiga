#include "hip/hip_runtime.h"
__device__ double device_linear_interpolate(double *x_vector, int x_length, double *y_vector, int y_length, double x_value){
    int i;       
    for (i=1; (i<x_length) && (x_vector[i-1]>x_value); i++);    
    if(i>1){--i;}else{i=1;}    
    return y_vector[i] - (y_vector[i]-y_vector[i-1])*(x_value-x_vector[i-1])/(x_vector[i]-x_vector[i-1]);
}

__device__ void generate_coords(double beam_diameter, double **position_all, double **speed_all, double eperm, int *prof_size, double *prof_r, double *prof_d, double *profx_r, double *profx_d){
    // thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int i;
    double Vabs, ionisation_yeald, xsec_rad, xsec_ang;
    double XR, XZ, XT;
    
    Vabs = sqrt(2 * eperm);
    
    /* cross section normalisation */
    if (prof_size[1] > 0){
        for (i=0; i<prof_size[1]; i++){
            profx_d[i] /= profx_r[i];
        }
    }
    
    /* initialize random generator */
    hiprandState state;
    hiprand_init(1234, idx, 0, state);
    
    /* set position of particles */
    do{
        ionisation_yeald = hiprand_uniform_double(state);
        XR =  device_linear_interpolate(prof_d, prof_size[0], prof_r, prof_size[0], ionisation_yeald);
        position_all[0][idx] = XR;
    }while (isnan(XR)||XR<0);
    do{
        //if (prof_size[1] <= 0){
            XZ = hiprand_uniform_double(state)*beam_diameter;
            XT = hiprand_uniform_double(state)*beam_diameter;
            position_all[1][idx] = XZ;
            position_all[2][idx] = XT;
        /*}else{
            ionisation_yeald = hiprand_uniform_double(state);
            xsec_ang = hiprand_uniform_double(state)*2*PI;
            xsec_rad = linear_interpolate(profx_d, prof_size[1], profx_r, prof_size[1], ionisation_yeald)*(beam_diameter/2);
            XZ[i]= sin(xsec_ang) * xsec_rad;
            XT[i]= cos(xsec_ang) * xsec_rad;
        }*/
    }while ((XZ*XZ+XT*XT)>=(beam_diameter/2)*(beam_diameter/2));
    
    // toroidal deflection 
    //position_all[2][idx] += tan(beam.toroidal_deflection) * ($R_defl - XR[i]);
    
    // set velocity of particles
    speed_all[0][idx] = -Vabs;//*cos(beam.vertical_deflection)*cos(beam.toroidal_deflection);
    speed_all[1][idx] =  0;//Vabs*sin(beam.vertical_deflection);
    speed_all[2][idx] =  0;//Vabs*cos(beam.vertical_deflection)*sin(beam.toroidal_deflection);
}
