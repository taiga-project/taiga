#include "hip/hip_runtime.h"
#define SPINE_INDEX_ERROR -1

__device__ void copy_local_field(double *r_grid, int NR, double *z_grid, int NZ, double position_rad, double position_z, int *local_spline_indices, double *local_spline_brad, double *local_spline_bz, double *local_spline_btor,  double **spline_brad, double **spline_bz, double **spline_btor){
    int rci, zci;
    int i, i2;
    
    for(rci=0;(r_grid[rci+1]<position_rad)&&(rci<NR-1);rci++){;}
    
    for(zci=0;(z_grid[zci+1]<position_z)&&(zci<NR-1);zci++){;}
    
    // Particle leave out the cell
    if ((local_spline_indices[0]!=rci)||(local_spline_indices[1]!=zci)){
        local_spline_indices[0] = rci;
        local_spline_indices[1] = zci;
    
        for(i=0;i<16;i++){
            i2 = (local_spline_indices[0])*(NZ-1)+local_spline_indices[1];
            local_spline_brad[i] = spline_brad[i][i2];
            local_spline_bz[i]   = spline_bz[i][i2];
            local_spline_btor[i] = spline_btor[i][i2];
        }
    }
}

__device__ void copy_local_field(double *r_grid, int NR, double *z_grid, int NZ, double position_rad, double position_z, int *local_spline_indices, double *local_spline_brad, double *local_spline_bz, double *local_spline_btor,  double **spline_brad, double **spline_bz, double **spline_btor,
                                double *local_spline_erad, double *local_spline_ez, double *local_spline_etor,  double **spline_erad, double **spline_ez, double **spline_etor){
    int rci, zci;
    int i, i2;
    
    for(rci=0;(r_grid[rci+1]<position_rad)&&(rci<NR-1);rci++){;}
    
    for(zci=0;(z_grid[zci+1]<position_z)&&(zci<NR-1);zci++){;}
    
    // Particle leave out the cell
    if ((local_spline_indices[0] != rci) || (local_spline_indices[1] != zci)){
        local_spline_indices[0] = rci;
        local_spline_indices[1] = zci;
    
        for(i=0;i<16;i++){
            i2 = (local_spline_indices[0])*(NZ-1)+local_spline_indices[1];
            local_spline_brad[i] = spline_brad[i][i2];
            local_spline_bz[i]   = spline_bz[i][i2];
            local_spline_btor[i] = spline_btor[i][i2];
            local_spline_erad[i] = spline_erad[i][i2];
            local_spline_ez[i]   = spline_ez[i][i2];
            local_spline_etor[i] = spline_etor[i][i2];
        }
    }
}

__device__ double calculate_local_field(double *local_spline, double dr, double dz){

    /* MATLAB CODE:
    sample2(3) =c11(bs1,bs2)*dsx^3*dsy^3 + c12(bs1,bs2)*dsx^3*dsy^2 + c13(bs1,bs2)*dsx^3*dsy + c14(bs1,bs2)*dsx^3 + ...
                c21(bs1,bs2)*dsx^2*dsy^3 + c22(bs1,bs2)*dsx^3*dsy^2 + c23(bs1,bs2)*dsx^2*dsy + c24(bs1,bs2)*dsx^2 + ...
                c31(bs1,bs2)*dsx  *dsy^3 + c32(bs1,bs2)*dsx  *dsy^2 + c33(bs1,bs2)*dsx  *dsy + c34(bs1,bs2)*dsx    + ...
                c41(bs1,bs2)      *dsy^3 + c42(bs1,bs2)      *dsy^2 + c43(bs1,bs2)      *dsy + c44(bs1,bs2);*/

    double local_field = 0.0, local_field_comp[16] ;
    for(int i=0;i<4;i++){
        for(int j=0;j<4;j++){
            local_field_comp[i*4+j] = local_spline[i*4+j]*pow(dr,3-i)*pow(dz,3-j);
        }
    }   
    
    for(int i=0;i<4;i++){
        for(int j=0;j<4;j++){
            local_field += local_field_comp[i*4+j];
        }
    }
    return local_field;
}


__device__ int traj(double *r_grid, int NR, double *z_grid, int NZ, double *position, double *speed, double **spline_brad, double **spline_bz, double **spline_btor, double eperm, double *detector_geometry, int N_step, int local_detcellid){

    // next grid
    int local_spline_indices[2];
    local_spline_indices[0] = SPINE_INDEX_ERROR;
    local_spline_indices[1] = SPINE_INDEX_ERROR;
        
    double local_spline_brad[16];
    double local_spline_bz[16];
    double local_spline_btor[16];
    
    double local_brad=0,local_bz,local_btor;
    double dr,dz;
    double R;
    
    double X[6], X_prev[6];
    
    int finished = local_detcellid + 1;

    X[0] = position[0];
    X[1] = position[1];
    X[2] = position[2];
    
    X[3] = speed[0];
    X[4] = speed[1];
    X[5] = speed[2];
    
    int loopi;
    for (loopi=0; (loopi<N_step && (!finished)); loopi++){
        // Get local magnetic field

        R = cyl2tor_coord(X[0], X[2]);
        copy_local_field(r_grid, NR, z_grid, NZ, R, X[1], local_spline_indices, local_spline_brad, local_spline_bz, local_spline_btor, spline_brad, spline_bz, spline_btor);
        
        dr = R-r_grid[local_spline_indices[0]];
        dz = X[1]-z_grid[local_spline_indices[1]];
    
        local_brad = calculate_local_field(local_spline_brad,dr,dz);
        local_bz   = calculate_local_field(local_spline_bz,  dr,dz);
        local_btor = calculate_local_field(local_spline_btor,dr,dz);
        local_brad = cyl2tor_rad(local_brad, local_btor, X[0], X[2]);
        local_btor = cyl2tor_field(local_brad, local_btor, X[0], X[2]);

        // archive coordinates
        X_prev[0] = X[0];
        X_prev[1] = X[1];
        X_prev[2] = X[2];
        X_prev[3] = X[3];
        X_prev[4] = X[4];
        X_prev[5] = X[5];
    
        solve_diffeq(X, local_brad, local_bz, local_btor, eperm);     

        finished = calculate_detection_position(X, X_prev, detector_geometry);
    }
    
    position[0] = X[0];
    position[1] = X[1];
    position[2] = X[2];    
    speed[0] = X[3];
    speed[1] = X[4];
    speed[2] = X[5];
    
    if (finished){
        local_detcellid = 0;
    }
    
    return local_detcellid;
}

__device__ int traj(double *r_grid, int NR, double *z_grid, int NZ, double *position, double *speed, double **spline_brad, double **spline_bz, double **spline_btor, double **spline_erad, double **spline_ez, double **spline_etor, double eperm, double *detector_geometry, int N_step, int local_detcellid){

    // next grid
    int local_spline_indices[2];
    local_spline_indices[0] = SPINE_INDEX_ERROR;
    local_spline_indices[1] = SPINE_INDEX_ERROR;
    
    double local_spline_brad[16];
    double local_spline_bz[16];
    double local_spline_btor[16];

    double local_spline_erad[16];
    double local_spline_ez[16];
    double local_spline_etor[16];
    
    double local_brad=0,local_bz=0,local_btor=0;
    double local_erad=0,local_ez=0,local_etor=0;
    double dr, dz;
    double R;    

    double X[6], X_prev[6];
    
    int finished = local_detcellid + 1;

    X[0] = position[0];
    X[1] = position[1];
    X[2] = position[2];
    
    X[3] = speed[0];
    X[4] = speed[1];
    X[5] = speed[2];
    
    int loopi;
    for (loopi=0; (loopi<N_step && (!finished)); loopi++){
        // Get local magnetic field

        R = cyl2tor_coord(X[0], X[2]);
        copy_local_field(r_grid, NR, z_grid, NZ, R, X[1], local_spline_indices, local_spline_brad, local_spline_bz, local_spline_btor, spline_brad, spline_bz, spline_btor, local_spline_erad, local_spline_ez, local_spline_etor, spline_erad, spline_ez, spline_etor);
        
        dr = R-r_grid[local_spline_indices[0]];
        dz = X[1]-z_grid[local_spline_indices[1]];
    
        local_brad = calculate_local_field(local_spline_brad,dr,dz);
        local_bz   = calculate_local_field(local_spline_bz,  dr,dz);
        local_btor = calculate_local_field(local_spline_btor,dr,dz);
        local_brad = cyl2tor_rad(local_brad, local_btor, X[0], X[2]);
        local_btor = cyl2tor_field(local_brad, local_btor, X[0], X[2]);
        
        local_erad = calculate_local_field(local_spline_erad,dr,dz);
        local_ez   = calculate_local_field(local_spline_ez,  dr,dz);
        local_etor = calculate_local_field(local_spline_etor,dr,dz);
        local_erad = cyl2tor_rad(local_erad, local_etor, X[0], X[2]);
        local_etor = cyl2tor_field(local_erad, local_etor, X[0], X[2]);

        // archive coordinates
        X_prev[0] = X[0];
        X_prev[1] = X[1];
        X_prev[2] = X[2];
        X_prev[3] = X[3];
        X_prev[4] = X[4];
        X_prev[5] = X[5];
    
        solve_diffeq(X, local_brad, local_bz, local_btor, local_erad, local_ez, local_etor, eperm);      

        finished = calculate_detection_position(X, X_prev, detector_geometry);
    }
    
    position[0] = X[0];
    position[1] = X[1];
    position[2] = X[2];
    speed[0] = X[3];
    speed[1] = X[4];
    speed[2] = X[5];    

    if (finished){
        local_detcellid = 0;
    }

    return local_detcellid;
}
