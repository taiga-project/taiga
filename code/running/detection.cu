
#include <hip/hip_runtime.h>
__device__ double interpolate(double y1, double y2, double vy1, double vy2, double x, double x1, double x2, double x2_x1){ 
    return (x2-x)/x2_x1 * (y1 + (x-x1)*vy1) + (x-x1)/x2_x1 * (y2 + (x-x2)*vy2);
}

__device__ double interpolate(double y1, double y2, double vy1, double vy2, double x, double x1, double x2){
    return interpolate(y1, y2, vy1, vy2, x, x1, x2, x2-x1);
}

__device__ double interpolate(double y1, double y2, double x, double x1, double x2, double x2_x1){ 
    return (x2-x)/x2_x1*y1 + (x-x1)/x2_x1*y2;
}

__device__ double interpolate(double y1, double y2, double x, double x1, double x2){
    return interpolate(y1, y2, x, x1, x2, x2-x1);
}

__device__ int calculate_detection_position(double *X, double *X_prev, double *detector_geometry){

    int finished = 0;
    
    double detector_R   = detector_geometry[0];
    double detector_z   = detector_geometry[1];
    double detector_tan = detector_geometry[3];
    
    double detector_distance = (X[0]-detector_R) + detector_tan*(X[1]-detector_z);
    double detector_distance_prev = (X_prev[0]-detector_R) + detector_tan*(X_prev[1]-detector_z);
    
    if((detector_distance*detector_distance_prev<=0) && (X[3]>0)){

        double X_new[6];     
        double detector_distance_change = (detector_distance-detector_distance_prev);
        double v = sqrt(X[3]*X[3] + X[4]*X[4] +  X[5]*X[5]);
        double v_prev = sqrt(X_prev[3]*X_prev[3] + X_prev[4]*X_prev[4] +  X_prev[5]*X_prev[5]);
        
        X_new[0] = interpolate(X_prev[0], X[0], X_prev[3]/v_prev, X[3]/v, 0, detector_distance_prev, detector_distance, detector_distance_change);
        X_new[1] = interpolate(X_prev[1], X[1], X_prev[4]/v_prev, X[4]/v, 0, detector_distance_prev, detector_distance, detector_distance_change);
        X_new[2] = interpolate(X_prev[2], X[2], X_prev[5]/v_prev, X[5]/v, 0, detector_distance_prev, detector_distance, detector_distance_change);
        X_new[3] = interpolate(X_prev[3], X[3], 0, detector_distance_prev, detector_distance, detector_distance_change);
        X_new[4] = interpolate(X_prev[4], X[4], 0, detector_distance_prev, detector_distance, detector_distance_change);
        X_new[5] = interpolate(X_prev[5], X[5], 0, detector_distance_prev, detector_distance, detector_distance_change);
        
        X[0] = X_new[0];
        X[1] = X_new[1];
        X[2] = X_new[2];
        X[3] = X_new[3];
        X[4] = X_new[4];
        X[5] = X_new[5];
        
        finished = 1;
        
    }
    
    return finished;
}
