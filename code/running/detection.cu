
#include <hip/hip_runtime.h>
__device__ double interpolate(double y1, double y2, double x, double x1, double x2, double x2_x1){ 
    return (x2-x)/x2_x1*y1 + (x-x1)/x2_x1*y2;
}

__device__ double interpolate(double y1, double y2, double x, double x1, double x2){
    return interpolate(y1, y2, x, x1, x2, x2-x1);
}

__device__ int calculate_detection_position(double *X, double *X_prev, double *detector_geometry){
    int finished = 0;
    
    double detector_R   = detector_geometry[0];
    double detector_z   = detector_geometry[1];
    double detector_tan = detector_geometry[3];
    
    double detector_distance = (X[0]-detector_R) + detector_tan*(X[1]-detector_z);
    double detector_distance_prev = (X_prev[0]-detector_R) + detector_tan*(X_prev[1]-detector_z);
    
    if((detector_distance*detector_distance_prev<=0) && (X[3]>0)){
        double detector_cos = 1/sqrt(1+detector_tan*detector_tan);
        detector_distance *= detector_cos;
        detector_distance_prev *= detector_cos;
        
        double X_new[6];     
        double detector_distance_rate = -detector_distance_prev/(detector_distance-detector_distance_prev);
        
        double v = sqrt(X[3]*X[3] + X[4]*X[4] +  X[5]*X[5]);
        double v_prev = sqrt(X_prev[3]*X_prev[3] + X_prev[4]*X_prev[4] +  X_prev[5]*X_prev[5]);
        
        for (int i=0; i<6; ++i) X_new[i] = interpolate(X_prev[i], X[i], 0, detector_distance_prev, detector_distance);
        for (int i=0; i<6; ++i) X[i] = X_new[i];
        
        finished = 1;
    }
    
    return finished;
}
