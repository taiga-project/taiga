#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

void export_detector(DetectorProp* shared_detector, DetectorProp *device_detector, TaigaGlobals *shared_global, ShotProp shot, RunProp run){
    if (shared_detector->detector_module_on){
        int *host_counter;
        size_t size_counter = shared_detector->number_of_detector_cells * sizeof(int);
        
        host_counter = (int*)malloc(size_counter);
        hipMemcpy(host_counter, shared_detector->counter, size_counter, hipMemcpyDeviceToHost);
        export_data(host_counter, shared_detector->number_of_detector_cells, run.folder_out, run.runnumber, "detector", "cellcounter.dat", shared_detector->length_ygrid);
        
        if (!FASTMODE){
            size_t size_detcellid = run.block_size * run.block_number * sizeof(int);
            int* host_detcellid =(int*)malloc(size_detcellid);
            hipMemcpy(host_detcellid, shared_global->detcellid, size_detcellid, hipMemcpyDeviceToHost);
            export_data(host_detcellid, shared_global->particle_number, run.folder_out, run.runnumber, "detector", "cellid.dat");
        }
    }
    
    CopyFile(concat("input/detector/", shot.detector_mask, "/detx"), concat(run.folder_out,"/",run.runnumber,"/detector/detx"));
    
    CopyFile(concat("input/detector/", shot.detector_mask, "/dety"), concat(run.folder_out,"/",run.runnumber,"/detector/dety"));
}
